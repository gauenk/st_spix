#include "hip/hip_runtime.h"

/********************************************************************

      Run BASS using the propograted superpixel segs and params

********************************************************************/


// -- cpp imports --
#include <stdio.h>
#include "pch.h"

// -- "external" import --
#ifndef MY_SP_STRUCT
#define MY_SP_STRUCT
#include "../bass/share/my_sp_struct.h"
#endif

// -- utils --
#include "rgb2lab.h"
#include "init_utils.h"
#include "init_sparams.h"
// #include "simple_init_sparams.h"
#include "seg_utils.h"
#include "sparams_io.h"
// #include "simple_sparams_io.h"

// -- primary functions --
#include "prop_bass.h"
// #include "simple_split_merge.h"
#include "split_merge_prop.h"
// #include "update_prop_params.h"
// #include "update_prop_seg.h"
#include "update_params.h"
#include "update_seg.h"


// -- define --
#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)
#define THREADS_PER_BLOCK 512


/**********************************************************

             -=-=-=-=- Main Function -=-=-=-=-=-

***********************************************************/

__host__ int prop_bass(float* img, int* seg,
                       spix_params* sp_params, bool* border,
                       spix_helper* sp_helper, spix_helper_sm* sm_helper,
                       int* sm_seg1 ,int* sm_seg2, int* sm_pairs,
                       int niters, int niters_seg, int sm_start,
                       float sigma2_app, float potts, float alpha_hastings,
                       int nspix, int nspix_buffer,
                       int nbatch, int width, int height, int nftrs){

    // -- init --
    int count = 1;
    int npix = height * width;
    // int nspix_buffer = nspix * 45;
    int max_spix = nspix;

    // -- run splits --
    count = 0;
    for (int idx = 0; idx < 1; idx++) { // num of different splits; each only once.
      max_spix = run_split_prop(img, seg, border, sp_params,
                                sp_helper, sm_helper, sm_seg1, sm_seg2, sm_pairs,
                                alpha_hastings, sigma2_app, count, idx, max_spix,
                                npix,nbatch,width,height,nftrs,nspix_buffer);
      update_params(img, seg, sp_params, sp_helper, sigma2_app,
                    npix, nspix_buffer, nbatch, width, nftrs);
    }

    // -- refine --
    for (int idx = 0; idx < niters; idx++) {
      // -- Update Parameters with Previous SuperpixelParams as Prior --
      update_params(img, seg, sp_params, sp_helper, sigma2_app,
                    npix, nspix_buffer, nbatch, width, nftrs);

      // -- Update Segmentation --
      update_seg(img, seg, border, sp_params, niters_seg,
                 sigma2_app, potts, npix, nbatch, width, height, nftrs);
    }

    // // -- run merge --
    // for (int idx = 0; idx < 4; idx++) {
    //   run_merge_prop(img, seg, border, sp_params,
    //                  sp_helper, sm_helper, sm_seg1, sm_seg2, sm_pairs,
    //                  alpha_hastings, sigma2_app, count, idx, max_spix, nspix,
    //                  npix,nbatch,width,height,nftrs,nspix_buffer);
    // }

    // -- apply changes from merge --
    update_params(img, seg, sp_params, sp_helper, sigma2_app,
                  npix, nspix_buffer, nbatch, width, nftrs);
    // update_seg(img, seg, border, sp_params,
    //            niters_seg, sigma2_app, potts,
    //            npix, nbatch, width, height, nftrs);

    // -- relabel from previou spix -- ?
    // relabel_from_history(....);

    // -- final border [legacy code; idk why we keep it] --
    CudaFindBorderPixels_end(seg, border, npix, nbatch, width, height);
    return max_spix;
}


/**********************************************************

             -=-=-=-=- Python API  -=-=-=-=-=-

***********************************************************/

std::tuple<torch::Tensor,PySuperpixelParams>
run_prop_bass(const torch::Tensor img, const torch::Tensor spix,
              const PySuperpixelParams prior_params,
              int niters, int niters_seg, int sm_start,
              int sp_size, float pix_var_i, float potts, float alpha_hastings){

    // -- check --
    CHECK_INPUT(img);
    CHECK_INPUT(spix);
    CHECK_INPUT(prior_params.mu_app);
    CHECK_INPUT(prior_params.mu_shape);
    CHECK_INPUT(prior_params.sigma_shape);
    CHECK_INPUT(prior_params.logdet_sigma_shape);
    CHECK_INPUT(prior_params.counts);
    CHECK_INPUT(prior_params.prior_counts);

    // -- unpack --
    int nbatch = spix.size(0);
    int height = spix.size(1);
    int width = spix.size(2);
    int nftrs = img.size(3);
    int npix = height*width;
    int nspix = prior_params.ids.size(0);

    // -- allocate filled spix --
    auto options_i32 = torch::TensorOptions().dtype(torch::kInt32)
      .layout(torch::kStrided).device(spix.device());
    auto options_f64 = torch::TensorOptions().dtype(torch::kFloat64)
      .layout(torch::kStrided).device(spix.device());
    torch::Tensor filled_spix = spix.clone();
    assert(nbatch==1);

    // -- allocate memory --
    // int nspix_buffer = nspix*50;
    int nspix_buffer = nspix*10;
    const int sparam_size = sizeof(spix_params);
    const int helper_size = sizeof(spix_helper);
    bool* border = (bool*)easy_allocate(nbatch*npix,sizeof(bool));
    spix_params* sp_params = get_tensors_as_params(prior_params,sp_size,
                                                         npix,nspix,nspix_buffer);
    // spix_params* sp_params=(spix_params*)easy_allocate(nspix_buffer,sparam_size);
    spix_helper* sp_helper=(spix_helper*)easy_allocate(nspix_buffer,helper_size);

    // -- init sp params from past --
    // assert(rescales.size(0) == 4);// must be of size 4
    // float4 rescale; // marked for deletion
    // rescale.x = rescales[0].item<int>();
    // rescale.y = rescales[1].item<int>();
    // rescale.z = rescales[2].item<int>();
    // rescale.w = rescales[3].item<int>();
    //init_sp_params_from_past(sp_params,prior_sp_params,rescale,nspix,nspix_buffer,npix);

    // -- compute pixel (inverse) variance info --
    float pix_half = float(pix_var_i/2) * float(pix_var_i/2);
    float sigma2_app =  pix_var_i;//1.0/pix_half;
    // float pix_var = std::sqrt(1./(4*pix_ivar.x));

    // pix_var.x = 1.0/pix_half;
    // pix_var.y = 1.0/pix_half;
    // pix_var.z = 1.0/pix_half;
    // float logdet_pix_var = log(pix_half * pix_half * pix_half);

    // -- Get pointers --
    float* img_ptr = img.data<float>();
    int* filled_spix_ptr = filled_spix.data<int>();

    // -- split/merge memory --
    const int sm_helper_size = sizeof(spix_helper_sm);
    int* sm_seg1 = (int*)easy_allocate(npix,sizeof(int));
    int* sm_seg2 = (int*)easy_allocate(npix,sizeof(int));
    int* sm_pairs = (int*)easy_allocate(2*npix,sizeof(int));
    spix_helper_sm* sm_helper=(spix_helper_sm*)easy_allocate(nspix_buffer,sm_helper_size);

    // -- allocate larger memory for prior map --
    // int* prior_map_ptr = (int*)easy_allocate(nspix_buffer,sizeof(int));
    // hipMemset(prior_map_ptr, -1, nspix_buffer*sizeof(int));
    // hipMemcpy(prior_map_ptr,prior_map_r_ptr,
    //            init_map_size*sizeof(int),hipMemcpyDeviceToDevice);

    // -- init superpixel params --
    // float prior_sigma_app = float(pix_var_i/2) * float(pix_var_i/2);
    // init_sp_params_from_past(sp_params,prior_sp_params,prior_map_ptr,
    //                          rescale,nspix,nspix_buffer,npix);

    // -=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-
    //
    //                 Run BASS
    //
    // -=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-

    // fprintf(stdout,"max_spix: %d\n",max_spix);
    int max_spix = prop_bass(img_ptr,filled_spix_ptr,sp_params,
                             border,sp_helper, sm_helper, sm_seg1, sm_seg2, sm_pairs,
                             niters, niters_seg, sm_start, sigma2_app, potts,
                             alpha_hastings, nspix, nspix_buffer,
                             nbatch, width, height, nftrs);
    // fprintf(stdout,"max_spix: %d\n",max_spix);

    // -- ensure new superpixels are compactly added to previous superpixels --
    int prev_max_spix = prior_params.ids.size(0);
    // fprintf(stdout,"prev_max_spix: %d\n",prev_max_spix);
    max_spix = compactify_new_superpixels(filled_spix,sp_params,
                                          prev_max_spix,max_spix,npix);


    // -- get spixel parameters as tensors --
    auto unique_ids = std::get<0>(at::_unique(filled_spix));
    auto ids = unique_ids.data<int>();
    int num_ids = unique_ids.sizes()[0];
    PySuperpixelParams params = get_output_params(sp_params,prior_params,
                                                  ids, num_ids, max_spix);
    run_update_prior(spix,params); // shift estimates to prior information @ spix

    // -- free --
    hipFree(border);
    hipFree(sp_params);
    hipFree(sp_helper);
    hipFree(sm_helper);
    hipFree(sm_seg1);
    hipFree(sm_seg2);
    hipFree(sm_pairs);

    return std::make_tuple(filled_spix,params);
}

void init_prop_bass(py::module &m){
  m.def("prop_bass", &run_prop_bass,"run propogated bass");
}

