#include "hip/hip_runtime.h"

/*********************************************************************************

     - This code shifts another tensor according to the "spix" and "flow"
     - ... this likely could have been done in native pytorch but this is
       much faster and we don't need grad...

*********************************************************************************/


// -- cpp imports --
#include <stdio.h>
#include "pch.h"

// -- define --
#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)
#define THREADS_PER_BLOCK 512


/**********************************************************

             -=-=-=-=- Main Function -=-=-=-=-=-

***********************************************************/


__global__
void shift_tensor_kernel(int* spix, int* flow, 
                         float* in_tensor, float* out_tensor,
                         int* counts, bool* contributors, int npix, int nspix,
                         int nbatch, int height, int width, int nftrs){

  // -- get pixel index --
  int pix_idx = threadIdx.x + blockIdx.x*blockDim.x;
  int h_idx = pix_idx / width;
  int w_idx = pix_idx % width;
  if (pix_idx>=npix) return;
  int batch_idx = blockIdx.y;
  pix_idx = pix_idx + npix*batch_idx;
  
  // -- superpixel at sources --
  int spix_label = *(spix+pix_idx);
  if ((spix_label<0) or (spix_label>=nspix)){ return; }

  // -- flow at source --
  int flow_offset = 2*(nspix*batch_idx+spix_label);
  int flow_w = *(flow+flow_offset);
  int flow_h = *(flow+flow_offset+1);

  // -- dest index --
  int h_dest = h_idx+flow_h;
  int w_dest = w_idx+flow_w;

  // -- check boundary; skip if oob --
  bool valid_h = (0<=h_dest) and (h_dest<=(height-1));
  bool valid_w = (0<=w_dest) and (w_dest<=(width-1));
  bool valid = valid_h and valid_w;
  if (not valid){ return; }

  // -- write to destination --
  int dest_idx = h_dest * width + w_dest + npix*batch_idx;
  float* in_tensor_ptr = in_tensor+nftrs*pix_idx;
  float* out_tensor_ptr = out_tensor+nftrs*dest_idx;
  int* counts_ptr = counts+dest_idx;

  // -- atomic write --
  int prev_max = atomicAdd(counts_ptr,1);
  for (int fi = 0; fi < nftrs; fi++){
    atomicAdd(out_tensor_ptr+fi,*(in_tensor_ptr+fi));
  }

  // -- mark superpixel as contributing [for ordering] --
  bool* contrib_ptr = contributors+nspix*(dest_idx)+spix_label;
  *contrib_ptr = true;

}



/**********************************************************

             -=-=-=-=- Python API  -=-=-=-=-=-

***********************************************************/

std::tuple<torch::Tensor,torch::Tensor,torch::Tensor>
run_shift_tensor(const torch::Tensor in_tensor,
                 const torch::Tensor spix,
                 const torch::Tensor flow){

    // -- check --
    CHECK_INPUT(in_tensor);
    CHECK_INPUT(spix);
    CHECK_INPUT(flow);

    // -- unpack --
    int nbatch = spix.size(0);
    int height = spix.size(1);
    int width = spix.size(2);
    int nftrs = in_tensor.size(3);
    int npix = height*width;
    int nspix = spix.max().item<int>()+1;
    int nspix_ = flow.size(1);
    assert(nspix == nspix_);

    // -- allocate filled spix --
    auto options_b = torch::TensorOptions().dtype(torch::kBool)
      .layout(torch::kStrided).device(spix.device());
    auto options_i32 = torch::TensorOptions().dtype(torch::kInt32)
      .layout(torch::kStrided).device(spix.device());
    auto options_f32 = torch::TensorOptions().dtype(torch::kFloat32)
      .layout(torch::kStrided).device(spix.device());

    // -- allocate memory --
    torch::Tensor out_tensor = torch::zeros({nbatch,height,width,nftrs},options_f32);
    torch::Tensor counts = torch::zeros({nbatch,height,width},options_i32);
    torch::Tensor contributors = torch::zeros({nbatch,height,width,nspix},options_b);

    // -- init launch info --
    int nblocks_for_npix = ceil( double(npix) / double(THREADS_PER_BLOCK) ); 
    dim3 BlocksPixels(nblocks_for_npix,nbatch);
    dim3 NumThreads(THREADS_PER_BLOCK,1);

    // -- run kernel --
    shift_tensor_kernel<<<BlocksPixels,NumThreads>>>(spix.data<int>(),flow.data<int>(),
                                                     in_tensor.data<float>(),
                                                     out_tensor.data<float>(),
                                                     counts.data<int>(),
                                                     contributors.data<bool>(),
                                                     npix,nspix,nbatch,
                                                     height,width,nftrs);

    return std::make_tuple(out_tensor,counts,contributors);
}

void init_shift_tensor(py::module &m){
  m.def("shift_tensor", &run_shift_tensor,"shift tensor");
}


