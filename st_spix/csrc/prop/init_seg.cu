#include "hip/hip_runtime.h"

#include "init_seg.h"
#define THREADS_PER_BLOCK 512

/*************************************************

              Initialize Superpixels

**************************************************/

__host__ int init_seg(int* seg, int sp_size, int width, int height, int nbatch){

  // -- superpixel info --
  int npix = height * width;
  double H = sqrt( double(pow(sp_size, 2)) / (1.5 *sqrt(3.0)) );
  double w = sqrt(3.0) * H;
  int max_num_sp_x = (int) floor(double(width)/w) + 1;
  int max_num_sp_y = (int) floor(double(height)/(1.5*H)) + 1;
  int nspix = max_num_sp_x * max_num_sp_y * 4; //Roy -Change

  // -- launch params --
  dim3 ThreadPerBlock(THREADS_PER_BLOCK,1);
  int nblocks_spix =  ceil(double(nspix) /double(THREADS_PER_BLOCK));
  dim3 BlockPerGrid_spix(nblocks_spix,nbatch);
  int nblocks_pix =  ceil(double(npix) /double(THREADS_PER_BLOCK));
  dim3 BlockPerGrid_pix(nblocks_pix,nbatch);
  double* centers;
  hipMalloc((void**) &centers, 2*nspix*sizeof(double));
  InitHexCenter<<<BlockPerGrid_spix,ThreadPerBlock>>>(centers, H, w, nspix,
                                                      max_num_sp_x, width, height); 
  InitHexSeg<<<BlockPerGrid_pix,ThreadPerBlock>>>(seg, centers,
                                                  nspix, npix, width);
  hipFree(centers);
  return nspix;

}

__global__ void InitHexCenter(double* centers, double H, double w, int npix,
                              int max_num_sp_x, int xdim, int ydim){
	int idx = threadIdx.x + blockIdx.x * blockDim.x; 
	if (idx >= npix) return;
    int x = idx % max_num_sp_x; 
    int y = idx / max_num_sp_x; 
    double xx = double(x) * w;
    double yy = double(y) * 1.5 *H; 
    if (y%2 == 0){
        xx = xx + 0.5*w;
    }
    centers[2*idx]  = xx;
    centers[2*idx+1]  = yy;    
}

__global__ void InitHexSeg(int* seg, double* centers,
                           int K, int npix, int xdim){
	int idx = threadIdx.x + blockIdx.x * blockDim.x; 	
	if (idx >= npix) return;
    int x = idx % xdim;
    int y = idx / xdim;   
    double dx,dy,d2;
    double D2 = DBL_MAX; 
    for (int j=0; j < K;  j++){
        dx = (x - centers[j*2+0]);
        dy = (y - centers[j*2+1]);
        d2 = dx*dx + dy*dy;
        if ( d2 <= D2){
              D2 = d2;  
              seg[idx]=j+1;
        }           
    } 
    return;	
}
