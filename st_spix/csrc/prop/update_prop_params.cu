#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#define THREADS_PER_BLOCK 1024
#include "update_prop_params.h"
#include <math.h>

/**********************************************************

             -=-=-=-=- Main Function -=-=-=-=-=-

***********************************************************/

__host__ void update_prop_params(const float* img, const int* spix,
                                 superpixel_params* sp_params,
                                 superpixel_GPU_helper* sp_gpu_helper,
                                 int* prev_means, int* prev_spix,
                                 const int npixels, const int nspix,
                                 const int nspix_buffer, const int nbatch,
                                 const int xdim, const int ydim, const int nftrs){

  	dim3 ThreadPerBlock(THREADS_PER_BLOCK,1);
    int num_block1 = ceil( double(npixels) / double(THREADS_PER_BLOCK) ); 
	int num_block2 = ceil( double(nspix_buffer) / double(THREADS_PER_BLOCK) );
    dim3 BlockPerGrid1(num_block1,nbatch);
    dim3 BlockPerGrid2(num_block2,nbatch);
    clear_fields<<<BlockPerGrid2,ThreadPerBlock>>>(sp_params,sp_gpu_helper,
                                                   nspix,nspix_buffer,nftrs);
	hipMemset(sp_gpu_helper, 0, nspix_buffer*sizeof(superpixel_GPU_helper));
    sum_by_label<<<BlockPerGrid1,ThreadPerBlock>>>(img,spix,sp_params,sp_gpu_helper,
                                                   npixels,nbatch,xdim,nftrs);
	calculate_mu_and_sigma<<<BlockPerGrid2,ThreadPerBlock>>>(\
     sp_params, sp_gpu_helper, prev_means, prev_spix, nspix, nspix_buffer); 

}

__global__
void clear_fields(superpixel_params* sp_params,
                  superpixel_GPU_helper* sp_gpu_helper,
                  const int nsuperpixel,
                  const int nsuperpixel_buffer,
                  const int nftrs){

	int k = threadIdx.x + blockIdx.x * blockDim.x;  // the label
	if (k>=nsuperpixel_buffer) return;
	if (sp_params[k].valid == 0) return;

	sp_params[k].count = 0;
	sp_params[k].log_count = 0.1;
	
	float3 mu_i;
	mu_i.x = 0;
	mu_i.y = 0;
	mu_i.z = 0;
	sp_params[k].mu_i = mu_i;

	double2 mu_s;
	mu_s.x = 0;
	mu_s.y = 0;
	sp_params[k].mu_s = mu_s;
}


__global__
void sum_by_label(const float* img,
                  const int* spix, superpixel_params* sp_params,
                  superpixel_GPU_helper* sp_gpu_helper,
                  const int npixels, const int nbatch,
                  const int xdim, const int nftrs) {
    // todo -- add nbatch and nftrs
    // getting the index of the pixel
    int t = threadIdx.x + blockIdx.x * blockDim.x;
	if (t>=npixels) return;

	//get the label
	int k = spix[t];
    if (k == -1){ return; } // invalid label

	atomicAdd(&sp_params[k].count, 1);
	atomicAdd(&sp_gpu_helper[k].mu_i_sum.x, img[3*t]);
	atomicAdd(&sp_gpu_helper[k].mu_i_sum.y, img[3*t+1]);
	atomicAdd(&sp_gpu_helper[k].mu_i_sum.z, img[3*t+2]);


	int x = t % xdim;
	int y = t / xdim; 
	int xx = x * x;
	int xy = x * y;
	int yy = y * y;

	atomicAdd(&sp_gpu_helper[k].mu_s_sum.x, x);
	atomicAdd(&sp_gpu_helper[k].mu_s_sum.y, y);
    atomicAdd((unsigned long long *)&sp_gpu_helper[k].sigma_s_sum.x, xx);
	atomicAdd((unsigned long long *)&sp_gpu_helper[k].sigma_s_sum.y, xy);
	atomicAdd((unsigned long long *)&sp_gpu_helper[k].sigma_s_sum.z, yy);
	
}



__global__
void calculate_mu_and_sigma(superpixel_params*  sp_params,
                            superpixel_GPU_helper* sp_gpu_helper,
                            int* prev_means, int* prev_spix,
                            const int nsuperpixel, const int nsuperpixel_buffer) {

    // -- update thread --
	int k = threadIdx.x + blockIdx.x * blockDim.x; // the label
	if (k>=nsuperpixel_buffer) return;
	if (sp_params[k].valid == 0) return;
    
    // -- read previou spix info --
    int prev_k = prev_spix[k];
    // int* means_prev_s = means_prev + prev_k*5;
    // int prev_mu_i_x = means_prev_s[0];
    // int prev_mu_i_y = means_prev_s[1];
    // int prev_mu_i_z = means_prev_s[1];
    // int prev_mu_s_x = means_prev_s[2];
    // int prev_mu_s_y = means_prev_s[3];

    // -- read curr --
	int count_int = sp_params[k].count;
	float a_prior = sp_params[k].prior_count;
	float prior_sigma_s_2 = a_prior * a_prior;
	double count = count_int * 1.0;
	double mu_x = 0.0;
	double mu_y = 0.0;

	// -- calculate the mean --
	if (count_int>0){

		sp_params[k].log_count = log(count);
	    mu_x = sp_gpu_helper[k].mu_s_sum.x / count;
	    mu_y = sp_gpu_helper[k].mu_s_sum.y / count;
		sp_params[k].mu_s.x = mu_x;
	    sp_params[k].mu_s.y = mu_y;
        
	    sp_params[k].mu_i.x = sp_gpu_helper[k].mu_i_sum.x / count;
		sp_params[k].mu_i.y = sp_gpu_helper[k].mu_i_sum.y / count;
  		sp_params[k].mu_i.z = sp_gpu_helper[k].mu_i_sum.z / count;

	}

	// -- calculate the covariance --
	double C00 = sp_gpu_helper[k].sigma_s_sum.x;
	double C01 = sp_gpu_helper[k].sigma_s_sum.y;
	double C11 = sp_gpu_helper[k].sigma_s_sum.z; 
	double total_count = (double) sp_params[k].count + a_prior;
	if (count_int > 3){
	    C00 = C00 - mu_x * mu_x * count;
	    C01 = C01 - mu_x * mu_y * count;
	    C11 = C11 - mu_y * mu_y * count;
	}

    // -- invert cov matrix --
    C00 = (prior_sigma_s_2 + C00) / (total_count - 3.0);
    C01 = C01 / (total_count - 3);
    C11 = (prior_sigma_s_2 + C11) / (total_count - 3.0);

    double detC = C00 * C11 - C01 * C01;
    if (detC <= 0){
      C00 = C00 + 0.00001;
      C11 = C11 + 0.00001;
      detC = C00*C11-C01*C01;
      if(detC <=0) detC = 0.0001;//hack
    }

    // -- finish-up inverse cov --
    sp_params[k].sigma_s.x = C11 / detC;     
    sp_params[k].sigma_s.y = -C01 / detC; 
    sp_params[k].sigma_s.z = C00 / detC; 
    sp_params[k].logdet_Sigma_s = log(detC);

}
