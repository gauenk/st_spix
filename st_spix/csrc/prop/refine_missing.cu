#include "hip/hip_runtime.h"

/*******************************************************

      - This finds a likely superpixel state
      after missing pixels are filled-in using "filled.cu",
      and after splitting with "split_disconnected.du".

      - This is necessary, because the "filled" superpixels are
      not in a likely state after the shift. The missing pixels have
      merely been assigned their spatial neighbor. This section of
      code actually runs BASS using the posterior of the parameter estimates.

      - This section of the code is different from "XXXX.cu"
      because updates can only effect the "missing" region.

*******************************************************/

// -- cpp imports --
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cuda/std/type_traits>
#include <torch/types.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>

// -- "external" import --
#ifndef MY_SP_STRUCT
#define MY_SP_STRUCT
#include "../bass/share/my_sp_struct.h"
#endif

// -- local import --
#include "rgb2lab.h"
#include "init_utils.h"
#include "seg_utils.h"
#include "refine_missing.h"
#include "update_prop_params.h"
#include "update_missing_seg.h"
#include "sparams_io.h"

// -- define --
#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)
#define THREADS_PER_BLOCK 512


/**********************************************************

             -=-=-=-=- Main Function -=-=-=-=-=-

***********************************************************/

__host__ void refine_missing(float* img, int* seg,
                             superpixel_params* sp_params,
                             superpixel_params* prior_params,
                             int* prior_map, bool* missing, bool* border,
                             superpixel_GPU_helper* sp_helper,
                             int niters, int niters_seg,
                             float3 pix_cov,float logdet_pix_cov,float potts,
                             int nspix, int nbatch, int width, int height, int nftrs){

    // -- init --
    int npix = height * width;
    int nspix_buffer = nspix * 45;
    for (int i = 0; i < niters; i++) {

      // -- Update Parameters with Previous SuperpixelParams as Prior --
      update_prop_params(img, seg, sp_params, sp_helper,
                         prior_params, prior_map, npix, nspix,
                         nspix_buffer, nbatch, width, height, nftrs);

      // -- Update Segmentation ONLY within missing pix --
      update_missing_seg(img, seg, border, missing, sp_params,
                         niters_seg, pix_cov, logdet_pix_cov, potts,
                         npix, nspix, nbatch, width, height, nftrs);

    }

    CudaFindBorderPixels_end(seg, border, npix, nbatch, width, height);
}


/**********************************************************

             -=-=-=-=- Python API  -=-=-=-=-=-

***********************************************************/

// torch::Tensor
std::tuple<torch::Tensor,PySuperpixelParams>
run_refine_missing(const torch::Tensor img_rgb,
                   const torch::Tensor spix,
                   const torch::Tensor missing,
                   const PySuperpixelParams prior_params,
                   const torch::Tensor prior_map,
                   int nspix, int niters, int niters_seg,
                   int sp_size, float pix_cov_i, float potts){

    // -- check --
    CHECK_INPUT(img_rgb);
    CHECK_INPUT(spix);
    CHECK_INPUT(missing);
    CHECK_INPUT(prior_params.mu_i);
    CHECK_INPUT(prior_params.mu_s);
    CHECK_INPUT(prior_params.sigma_s);
    CHECK_INPUT(prior_params.logdet_Sigma_s);
    CHECK_INPUT(prior_params.counts);
    CHECK_INPUT(prior_params.prior_counts);
    CHECK_INPUT(prior_map);

    // -- unpack --
    int nbatch = spix.size(0);
    int height = spix.size(1);
    int width = spix.size(2);
    int nftrs = img_rgb.size(3);
    int npix = height*width;
    int nmissing = missing.sum().item<int>();
    int init_map_size = prior_map.size(0);
    
    // -- allocate filled spix --
    auto options_i32 = torch::TensorOptions().dtype(torch::kInt32)
      .layout(torch::kStrided).device(spix.device());
    torch::Tensor filled_spix = spix.clone();
    assert(nbatch==1);

    // -- allocate memory --
    int nspix_buffer = nspix*50;
    const int sparam_size = sizeof(superpixel_params);
    const int helper_size = sizeof(superpixel_GPU_helper);
    bool* border = (bool*)easy_allocate(nbatch*npix,sizeof(bool));
    superpixel_params* prior_sp_params = get_tensors_as_params(prior_params,sp_size,
                                                               npix,nspix,nspix_buffer);
    superpixel_params* sp_params=(superpixel_params*)easy_allocate(nspix_buffer,
                                                                   sparam_size);
    superpixel_GPU_helper* sp_helper=(superpixel_GPU_helper*)easy_allocate(nspix_buffer,
                                                                           helper_size);
    init_sp_params(sp_params,sp_size,nspix,nspix_buffer,npix);

    // bool* border = allocate_border(nbatch*npix);
    // superpixel_params* sp_params = allocate_sp_params(nspix_buffer);
    // superpixel_GPU_helper* sp_helper = allocate_sp_helper(nspix_buffer);
    // init_sp_params(sp_params,sp_size,nspix,nspix_buffer,npix);

    // -- compute pixel (inverse) covariance info --
    float pix_half = float(pix_cov_i/2) * float(pix_cov_i/2);
    float3 pix_cov;
    pix_cov.x = 1.0/pix_half;
    pix_cov.y = 1.0/pix_half;
    pix_cov.z = 1.0/pix_half;
    float logdet_pix_cov = log(pix_half * pix_half * pix_half);

    // -- convert image color --
    auto img_lab = img_rgb.clone();
    rgb2lab(img_rgb.data<float>(),img_lab.data<float>(),npix,nbatch);

    // -- Get pointers --
    float* img_ptr = img_lab.data<float>();
    int* filled_spix_ptr = filled_spix.data<int>();
    bool* missing_ptr = missing.data<bool>();
    int* prior_map_r_ptr = prior_map.data<int>();

    // -- allocate larger memory for prior map --
    int* prior_map_ptr = (int*)easy_allocate(nspix_buffer,sizeof(int));
    hipMemset(prior_map_ptr, -1, nspix_buffer*sizeof(int));
    hipMemcpy(prior_map_ptr,prior_map_r_ptr,
               init_map_size*sizeof(int),hipMemcpyDeviceToDevice);

    // -- run fill --
    if (nmissing>0){
      refine_missing(img_ptr,filled_spix_ptr,sp_params,
                     prior_sp_params, prior_map_ptr, missing_ptr, border,
                     sp_helper, niters, niters_seg, pix_cov, logdet_pix_cov,
                     potts, nspix, nbatch, width, height, nftrs);
    }


    // -- get spixel parameters as tensors --
    auto unique_ids = std::get<0>(at::_unique(filled_spix));
    auto ids = unique_ids.data<int>();
    int nspix_post = unique_ids.sizes()[0];
    PySuperpixelParams params = get_params_as_tensors(sp_params,ids,nspix_post);

    // -- free --
    hipFree(border);
    hipFree(sp_params);
    hipFree(sp_helper);

    return std::make_tuple(filled_spix,params);
}

void init_refine_missing(py::module &m){
  m.def("refine_missing", &run_refine_missing,"refine missing labels");
}

