#include "hip/hip_runtime.h"

#include <string>
#include <iostream>
#include <fstream>
#include <sstream>
#include <float.h>
#include <math.h>

#include <stdio.h>
#ifndef WIN32
#include <unistd.h>
#endif

#ifndef OUT_OF_BOUNDS_LABEL
#define OUT_OF_BOUNDS_LABEL -1
#endif
#define THREADS_PER_BLOCK 512

#include "split_merge.h"


__host__
int run_split(const float* img, int* seg, bool* border,
              spix_params* sp_params, spix_helper* sp_helper,
              spix_helper_sm* sm_helper,
              int* sm_seg1 ,int* sm_seg2, int* sm_pairs,
              float alpha_hastings, float sigma2_app, float sigma2_size,
              int& count, int idx, int max_spix, 
              const int sp_size, 
              const int npix, const int nbatch,
              const int width, const int height,
              const int nftrs, const int nspix_buffer){

  if(idx%4 == 0){
    count += 1;
    int direction = count%2+1;
    // -- run split --
    max_spix = CudaCalcSplitCandidate(img, seg, border,
                                       sp_params, sp_helper, sm_helper,
                                       sm_seg1, sm_seg2, sm_pairs,
                                       sp_size,npix,nbatch,width,height,nftrs,
                                       nspix_buffer, max_spix,
                                       direction, alpha_hastings,
                                       sigma2_app, sigma2_size);

  }
  return max_spix;
}

__host__
void run_merge(const float* img, int* seg, bool* border,
               spix_params* sp_params, spix_helper* sp_helper,
               spix_helper_sm* sm_helper,
               int* sm_seg1, int* sm_seg2, int* sm_pairs,
               float alpha_hastings,
               float sigma2_app, float sigma2_size,
               int& count, int idx, int max_spix,
               const int sp_size, const int npix, const int nbatch,
               const int width, const int height,
               const int nftrs, const int nspix_buffer){

  if( idx%4 == 2){
    // -- run merge --
    int direction = count%2;
    // fprintf(stdout,"idx,count,direction: %d,%d,%d\n",idx,count,direction);
    CudaCalcMergeCandidate(img, seg, border,
                           sp_params, sp_helper, sm_helper, sm_pairs,
                           sp_size,npix,nbatch,width,height,nftrs,
                           nspix_buffer,direction, alpha_hastings,
                           sigma2_app, sigma2_size);

  }
}

__host__ void CudaCalcMergeCandidate(const float* img, int* seg, bool* border,
                                     spix_params* sp_params,spix_helper* sp_helper,
                                     spix_helper_sm* sm_helper,int* sm_pairs,
                                     const int sp_size,
                                     const int npix, const int nbatch,
                                     const int width, const int height,
                                     const int nftrs, const int nspix_buffer,
                                     const int direction, float log_alpha,
                                     float sigma2_app, float sigma2_size){

    int num_block = ceil( double(npix) / double(THREADS_PER_BLOCK) ); 
    int num_block2 = ceil( double(nspix_buffer) / double(THREADS_PER_BLOCK) );
    dim3 BlockPerGrid2(num_block2,nbatch);
    dim3 ThreadPerBlock(THREADS_PER_BLOCK,1);
    dim3 BlockPerGrid(num_block,nbatch);
    // float alpha_hasting_ratio = alpha;
    // float a_0 = 1e6;
    // float b_0 = sigma2_app * (a_0) ;

    int nvalid_cpu;
    int* nvalid;
    hipMalloc((void **)&nvalid, sizeof(int));
    hipMemset(nvalid, 0,sizeof(int));

    int nmerges;
    int* nmerges_gpu;
    hipMalloc((void **)&nmerges_gpu, sizeof(int));
    hipMemset(nmerges_gpu, 0,sizeof(int));

    init_sm<<<BlockPerGrid2,ThreadPerBlock>>>(img,seg,sp_params,sm_helper,
                                              nspix_buffer, nbatch, width,
                                              nftrs, npix, sm_pairs, nvalid);
    // fprintf(stdout,"direction: %d\n",direction);
    calc_merge_candidate<<<BlockPerGrid,ThreadPerBlock>>>(seg,border, sm_pairs,
                                                          npix, nbatch, width,
                                                          height, direction); 
    sum_by_label<<<BlockPerGrid,ThreadPerBlock>>>(img,seg,sp_params,sm_helper,
                                                        npix, nbatch, width,  nftrs);
    merge_marginal_likelihood<<<BlockPerGrid2,ThreadPerBlock>>>(sm_pairs,sp_params,
                                                                sm_helper,
                                                                sp_size, npix,
                                                                nbatch, width,
                                                                nspix_buffer,
                                                                sigma2_app,sigma2_size);
    merge_hastings_ratio<<<BlockPerGrid2,ThreadPerBlock>>>(img,  sm_pairs, sp_params,
                                                           sp_helper, sm_helper,
                                                           npix, nbatch, width,
                                                           nftrs, nspix_buffer,
                                                           log_alpha,nmerges_gpu);
    // -- count number of merges --
    hipMemcpy(&nmerges,nmerges_gpu, sizeof(int), hipMemcpyDeviceToHost);
    // printf("nmerges: %d\n",nmerges);
    hipMemset(nmerges_gpu, 0,sizeof(int));
    hipMemcpy(&nvalid_cpu, nvalid, sizeof(int), hipMemcpyDeviceToHost);
    // printf("[merge] nvalid: %d\n",nvalid_cpu);

    
    // -- actually merge --
    remove_sp<<<BlockPerGrid2,ThreadPerBlock>>>(sm_pairs,sp_params,
                                                sm_helper,nspix_buffer);
    merge_sp<<<BlockPerGrid,ThreadPerBlock>>>(seg,border, sm_pairs, sp_params,
                                              sm_helper, npix, nbatch, width, height);  

    // -- free! --
    hipFree(nvalid);
    hipFree(nmerges_gpu);


}



// void sumIntArray(int* data, int H, int W) {

//     auto options_i32 = torch::TensorOptions().dtype(torch::kInt32)
//       .layout(torch::kStrided).device("cuda");
//     torch::Tensor tensor = torch::from_blob(data, {H, W}, options_i32);
//     int sum = tensor.sum().item<int>();
//     printf("sum: %d\n",sum);
// }

// void saveIntArray(int* data, int H, int W, const std::string& filename) {
//     // Create a PyTorch tensor from the raw pointer
//     // Note: 'torch::kInt' specifies that the tensor will have an integer data type
//     auto options_i32 = torch::TensorOptions().dtype(torch::kInt32)
//       .layout(torch::kStrided).device("cuda");
//     torch::Tensor tensor = torch::from_blob(data, {H, W}, options_i32);

//     // Save the tensor to a file
//     torch::save(tensor, filename);
// }



__host__ int CudaCalcSplitCandidate(const float* img, int* seg, bool* border,
                                    spix_params* sp_params,
                                    spix_helper* sp_helper,
                                    spix_helper_sm* sm_helper,
                                    int* sm_seg1, int* sm_seg2, int* sm_pairs,
                                    const int sp_size,
                                    const int npix, const int nbatch, const int width,
                                    const int height, const int nftrs,
                                    const int nspix_buffer, int max_spix,
                                    int direction, float alpha,
                                    float sigma2_app, float sigma2_size){

    if (max_spix>nspix_buffer/2){ return max_spix; }
    int num_block = ceil( double(npix) / double(THREADS_PER_BLOCK) ); 
    // printf("nspix_buffer: %d\n",nspix_buffer);
    int num_block2 = ceil( double(nspix_buffer) / double(THREADS_PER_BLOCK) );
    dim3 BlockPerGrid2(num_block2,1);
    dim3 ThreadPerBlock(THREADS_PER_BLOCK,1);
    dim3 BlockPerGrid(num_block,1);
    float alpha_hasting_ratio =  alpha;
    // float a_0 = 1e6;
    // float b_0 = sigma2_app * (a_0) ;
    // float b_0;
    int* done_gpu;
    int* max_sp;
    int* nvalid;
    int nvalid_cpu;
    hipMalloc((void **)&nvalid, sizeof(int));
    hipMalloc((void **)&max_sp, sizeof(int));
    hipMalloc((void **)&done_gpu, sizeof(int)); 
    hipMemset(nvalid, 0,sizeof(int));

    // hipMemset(sm_seg1, 0, npix*sizeof(int));
    // hipMemset(sm_seg2, 0, npix*sizeof(int));
    hipMemset(sm_seg1, -1, npix*sizeof(int));
    hipMemset(sm_seg2, -1, npix*sizeof(int));

    init_sm<<<BlockPerGrid2,ThreadPerBlock>>>(img,seg,sp_params, sm_helper,
                                              nspix_buffer, nbatch, width,
                                              nftrs, npix, sm_pairs, nvalid);
    hipMemcpy(&nvalid_cpu, nvalid, sizeof(int), hipMemcpyDeviceToHost);
    // printf("[split] nvalid: %d\n",nvalid_cpu);
    hipMemset(nvalid, 0,sizeof(int));
    // printf("direction: %d\n",direction);

    init_split<<<BlockPerGrid2,ThreadPerBlock>>>(border,sm_seg1,sp_params,
                                                 nspix_buffer,
                                                 nbatch, width, height, direction,
                                                 seg, max_sp, max_spix);
    init_split<<<BlockPerGrid2,ThreadPerBlock>>>(border,sm_seg2,sp_params,
                                                 nspix_buffer,
                                                 nbatch, width,height, -direction,
                                                 seg, max_sp, max_spix);


    // -- compute sum of sm_seg2 --
    // sumIntArray(sm_seg1,height,width);
    // sumIntArray(sm_seg2,height,width);
    
    // idk what "split_sp" is doing here; init_sm clears the merge fields and
    // so the function returns immediately...
    split_sp<<<BlockPerGrid,ThreadPerBlock>>>(seg,sm_seg1,sm_pairs,
                                              sp_params, sm_helper, npix,
                                              nbatch, width, height, max_spix);

    // gpuErrchk( hipPeekAtLastError() );
    // gpuErrchk( hipDeviceSynchronize() );
    // printf("width,height,npix: %d,%d,%d\n",width,height,npix);

    int distance = 1;
    int done = 1;
    while(done)
    {
      // // -- debug REMOVE ME --
      //   if (distance < 10){
      //     char buffer[50];
      //     sprintf(buffer, "sm_spix1_%d",distance);
      //     std::string fn = buffer;
      //     saveIntArray(sm_seg1, height, width, fn);
      //   }


        hipMemset(done_gpu, 0, sizeof(int));
        // hipMemcpy(&done, done_gpu, sizeof(int), hipMemcpyDeviceToHost);
        calc_split_candidate<<<BlockPerGrid,ThreadPerBlock>>>(\
                 sm_seg1,seg,border,distance, done_gpu, npix, nbatch, width, height); 
        distance++;
        hipMemcpy(&done, done_gpu, sizeof(int), hipMemcpyDeviceToHost);
        // printf("[a] distance: %d\n",distance);

        // // -- debug REMOVE ME --
        // if (distance > 5000){
        //   // saveIntArray(int* data, int H, int W, const std::string& filename) 
        //   // saveIntArray(sm_seg1,height,width,spir"sm_seg1.pth");
        //   char buffer[50];
        //   sprintf(buffer, "sm_spix1_%d",distance);
        //   std::string fn = buffer;
        //   saveIntArray(sm_seg1, height, width, fn);

        //   if (distance > 5005){
        //     exit(1);
        //   }
        // }
        // gpuErrchk( hipPeekAtLastError() );
        // gpuErrchk( hipDeviceSynchronize() );
    }

    done = 1;
    distance = 1;
    while(done)
    {
		hipMemset(done_gpu, 0, sizeof(int));
        // hipMemcpy(&done, done_gpu, sizeof(int), hipMemcpyDeviceToHost);
        calc_split_candidate<<<BlockPerGrid,ThreadPerBlock>>>(\
                sm_seg2,seg,border,distance, done_gpu, npix, nbatch, width, height); 
        distance++;
        hipMemcpy(&done, done_gpu, sizeof(int), hipMemcpyDeviceToHost);
        // printf("[b] distance: %d\n",distance);
        // gpuErrchk( hipPeekAtLastError() );
        // gpuErrchk( hipDeviceSynchronize() );
    }

    // updates the segmentation to the two regions; split either left/right or up/down.
    // printf("max_spix: %d\n",max_spix);
    calc_seg_split<<<BlockPerGrid,ThreadPerBlock>>>(sm_seg1,sm_seg2,
                                                    seg, npix, nbatch, max_spix);
    // std::string fname_split1_post = "split1_post";
    // write_tensor_to_file_v2(sm_seg1,height,width,fname_split1_post);

    // computes summaries stats for each split
    // printf("npix: %d\n",npix);
    sum_by_label<<<BlockPerGrid,ThreadPerBlock>>>(img, sm_seg1, sp_params,
                                                        sm_helper, npix, nbatch,
                                                        width,nftrs);
    // gpuErrchk( hipPeekAtLastError() );
    // gpuErrchk( hipDeviceSynchronize() );
    // calc_bn_split<<<BlockPerGrid2,ThreadPerBlock>>>(sm_pairs, sp_params, sp_helper,
    //                                                 sm_helper, npix, nbatch, width,
    //                                                 nspix_buffer, b_0, max_spix);

    // gpuErrchk( hipPeekAtLastError() );
    // gpuErrchk( hipDeviceSynchronize() );
    // split_likelihood<<<BlockPerGrid2,ThreadPerBlock>>>(img,  sm_pairs,
    //                                                    sp_params,  sp_helper,
    //                                                    sm_helper,
    //                                                    npix, nbatch, width, nftrs,
    //                                                    nspix_buffer, a_0,
    //                                                    b_0, max_spix);

    split_marginal_likelihood<<<BlockPerGrid2,ThreadPerBlock>>>(\
        sp_params,sm_helper,sp_size,npix,nbatch,width,nspix_buffer,
        sigma2_app, sigma2_size, max_spix);

    // calc_marginal_likelihood<<<BlockPerGrid2,ThreadPerBlock>>>(\
    //     sp_params,sm_helper,npix,nbatch,width,nspix_buffer,
    //     sigma2_app, max_spix);

    // gpuErrchk( hipPeekAtLastError() );
    // gpuErrchk( hipDeviceSynchronize() );
    // fprintf(stdout,"[s_m.cu] max_spix: %d\n",max_spix);
    split_hastings_ratio<<<BlockPerGrid2,ThreadPerBlock>>>(img, sm_pairs, sp_params,
                                                           sp_helper, sm_helper,
                                                           npix, nbatch, width, nftrs,
                                                           nspix_buffer,sp_size,
                                                           alpha_hasting_ratio,
                                                           max_spix, max_sp);

    // -- do the split --
    split_sp<<<BlockPerGrid,ThreadPerBlock>>>(seg,sm_seg1,sm_pairs,
                                              sp_params, sm_helper, npix,
                                              nbatch, width, height, max_spix);


    // gpuErrchk( hipPeekAtLastError() );
    // gpuErrchk( hipDeviceSynchronize() );
    // -- nvalid --
    int prev_max_sp = max_spix;
    hipMemcpy(&max_spix, max_sp, sizeof(int), hipMemcpyDeviceToHost);
    // printf("[split] nsplits: %d\n",max_spix-prev_max_sp);

    // -- free --
    hipFree(nvalid);
    hipFree(max_sp);
    hipFree(done_gpu);

    return max_spix;
}


__global__ void init_sm(const float* img, const int* seg_gpu,
                        spix_params* sp_params,
                        spix_helper_sm* sm_helper,
                        const int nspix_buffer, const int nbatch,
                        const int width,const int nftrs, int npix,
                        int* sm_pairs, int* nvalid) {
	int k = threadIdx.x + blockIdx.x * blockDim.x;  // the label
	if (k>=nspix_buffer) return;
	// if (sp_params[k].valid == 0) return;
    // atomicAdd(nvalid,1); // update valid

	// if (sp_params[k].valid != 0) {
    //   atomicAdd(nvalid,1); // update valid
    // }


    sm_helper[k].b_n_app.x = 0;
    sm_helper[k].b_n_app.y = 0;
    sm_helper[k].b_n_app.z = 0;

	sm_helper[k].sq_sum_app.x = 0;
	sm_helper[k].sq_sum_app.y = 0;
	sm_helper[k].sq_sum_app.z = 0;
    sm_helper[k].sum_app.x = 0;
	sm_helper[k].sum_app.y = 0;
	sm_helper[k].sum_app.z = 0;
    sm_helper[k].count_f = 0;
    sm_helper[k].count = 0;
    sm_helper[k].hasting = -999999;
    //sp_params[k].count = 0;

    sm_helper[k].merge = false;
    sm_helper[k].remove = false;

    // -- invalidate --
    sm_pairs[2*k] = -1;
    sm_pairs[2*k+1] = -1;
    // int k2 = 2*k;
    // if (k2 < 2*npix){
    //   sm_pairs[k2] = -1;
    // }
    // if (k2+1 < 2*npix){
    //   sm_pairs[k2+1] = -1;
    // }

}
__global__
void merge_marginal_likelihood(int* sm_pairs, spix_params* sp_params,
                               spix_helper_sm* sm_helper,
                               const int sp_size,
                               const int npix, const int nbatch,
                               const int width, const int nspix_buffer,
                               float sigma2_app, float sigma2_size){

    /********************
           Init
    **********************/

    // -- init --
	int k = threadIdx.x + blockIdx.x * blockDim.x;  // the label
	if (k>=nspix_buffer) return;
	if (sp_params[k].valid == 0) return;
    int s = sm_pairs[2*k+1];
    if (s < 0){ return; }
    float count_s = __ldg(&sm_helper[s].count);
    float count_k = __ldg(&sm_helper[k].count);
    // float count_s = __ldg(&sp_params[s].count);
    // float count_k = __ldg(&sp_params[k].count);
    float count_f = count_s + count_k;

    if((count_f<1)||( count_k<1)||(count_s<1)) return;

    /********************
  
          Appearance
   
    **********************/

    double3 sum_s = sm_helper[s].sum_app;
    double3 sum_k = sm_helper[k].sum_app;
    double3 sum_f;
    sum_f.x = sum_s.x + sum_k.x;
    sum_f.y = sum_s.y + sum_k.y;
    sum_f.z = sum_s.z + sum_k.z;

    double3 sq_sum_s = sm_helper[s].sq_sum_app;
    double3 sq_sum_k = sm_helper[k].sq_sum_app;
    double3 sq_sum_f;
    sq_sum_f.x = sq_sum_s.x + sq_sum_k.x;
    sq_sum_f.y = sq_sum_s.y + sq_sum_k.y;
    sq_sum_f.z = sq_sum_s.z + sq_sum_k.z;

    // -- appearance --
    // double lprob_k = marginal_likelihood_app(sum_k,sq_sum_k,count_k,sigma2_app);
    // double lprob_s = marginal_likelihood_app(sum_s,sq_sum_s,count_s,sigma2_app);
    // double lprob_f = marginal_likelihood_app(sum_f,sq_sum_f,count_f,sigma2_app);
    double sigma2_prior_var = 1.;
    double lprob_k = appearance_variance(sum_k,sq_sum_k,count_k,sigma2_prior_var);
    double lprob_s = appearance_variance(sum_s,sq_sum_s,count_s,sigma2_prior_var);
    double lprob_f = appearance_variance(sum_f,sq_sum_f,count_f,sigma2_prior_var);


    // -- include size term --
    // int sp_size2 = sp_size*sp_size;
    lprob_k += size_likelihood(count_k,sp_size,sigma2_size);
    lprob_s += size_likelihood(count_s,sp_size,sigma2_size);
    lprob_f += size_likelihood(count_f,sp_size,sigma2_size);

    // -- include size term --
    // lprob_k += size_beta_likelihood(count_k,sp_size,sigma2_size,npix);
    // lprob_s += size_beta_likelihood(count_s,sp_size,sigma2_size,npix);
    // lprob_f += size_beta_likelihood(count_f,sp_size,sigma2_size,npix);

    // -- write --
    sm_helper[k].numerator_app = lprob_k;
    sm_helper[s].numerator_app = lprob_s;
    sm_helper[k].numerator_f_app = lprob_f;


}

__global__ void merge_hastings_ratio(const float* img, int* sm_pairs,
                                    spix_params* sp_params,
                                    spix_helper* sp_helper,
                                    spix_helper_sm* sm_helper,
                                    const int npix, const int nbatch, const int width,
                                    const int nftrs, const int nspix_buffer,
                                     float log_alpha, int* nmerges) {

	// getting the index of the pixel
	int k = threadIdx.x + blockIdx.x * blockDim.x;  // the label
	if (k>=nspix_buffer) return;
	if (sp_params[k].valid == 0) return;
    int s = sm_pairs[2*k+1];
    if(s<0) return;
	if (sp_params[s].valid == 0) return;
    // if(s<=0) return;

    // -- unpack --
    float count_s = __ldg(&sm_helper[s].count);
    float count_k = __ldg(&sm_helper[k].count);
    // float count_s = __ldg(&sp_params[s].count);
    // float count_k = __ldg(&sp_params[k].count);
    float count_f = count_s + count_k;
    if((count_f<1)||(count_k<1)||(count_s<1)) return;
    float lprob_k = __ldg(&sm_helper[k].numerator_app);
    float lprob_s = __ldg(&sm_helper[s].numerator_app);
    float lprob_f = __ldg(&sm_helper[k].numerator_f_app);

    // -- compute hastings --
    double alpha = exp(log_alpha);
    // double log_const = lgammaf(count_f) + lgammaf(alpha) \
    //   + lgammaf(alpha / 2 + count_k) + lgammaf(alpha / 2 + count_s)\
    //   - lgammaf(count_s) - lgammaf(count_k) - lgammaf(alpha+count_f)-2*lgamma(alpha/2);
    double log_const = 0;
    double hastings = log_const + lprob_f - lprob_k - lprob_s - log_alpha;
    // double hastings = lprob_f - lprob_k - lprob_s - log_alpha;
    sm_helper[k].hasting = hastings;
    // sm_helper[k].merge = hastings > 0;
    // sm_helper[s].merge = hastings > 0;

    // printf("info[%d,%d] %f,%f,%f|%lf,%f,%f,%f,%lf|\n",
    //        k,s,count_s,count_k,count_f,
    //        log_const,lprob_f,lprob_k,lprob_s,hastings);

    // -- Check hastings and update --
    if(hastings > 0){

      // printf("info[%d,%d] %f,%f,%f|%lf,%f,%f,%f,%lf|\n",k,s,
      //        count_s,count_k,count_f,
      //        log_const,lprob_f,lprob_k,lprob_s,hastings);

      // printf("info[%d,%d] %lf,%f,%f,%f\n",k,s,log_const,lprob_f,lprob_k,lprob_s);
      int curr_max = atomicMax(&sm_pairs[2*s],k);
      if( curr_max == -1){
        atomicAdd(nmerges,1);
        sm_helper[k].merge = true;
      // }else{
      //   sm_pairs[2*s] = curr_max;
      }

    }
    return;
}




__global__
void split_marginal_likelihood(spix_params* sp_params,
                               spix_helper_sm* sm_helper,
                               const int sp_size,
                               const int npix, const int nbatch,
                               const int width, const int nspix_buffer,
                               float sigma2_app, float sigma2_size, int max_spix){

    /********************
           Init
    **********************/

    // -- init --
	int k = threadIdx.x + blockIdx.x * blockDim.x;  // the label
	if (k>=nspix_buffer) return;
	if (sp_params[k].valid == 0) return;

    // -- split --
    int s = k + (max_spix+1);
	if (s>=nspix_buffer) return;
    // int count_f = __ldg(&sp_params[k].count);
    int count_k = __ldg(&sm_helper[k].count);
    int count_s = __ldg(&sm_helper[s].count);
    int count_f = count_k + count_s;

    // printf("[%d]: %d,%d,%d\n",k,count_k,count_s,count_f);
    if((count_f<1)||( count_k<1)||(count_s<1)) return;

    /********************
  
          Appearance
   
    **********************/

    // float3 mu_pr_k = sp_params[k].prior_mu_app;
    // float3 mu_pr_f = mu_pr_k;
    // sp_params[s].prior_mu_app.x = 0;
    // sp_params[s].prior_mu_app.y = 0;
    // sp_params[s].prior_mu_app.z = 0;
    // float3 mu_pr_s = sp_params[s].prior_mu_app;

    // sp_params[s].prior_mu_app_count = 1;
    // int prior_mu_app_count_s = sp_params[s].prior_mu_app_count;
    // int prior_mu_app_count_k = sp_params[k].prior_mu_app_count;
    // int prior_mu_app_count_f = prior_mu_app_count_k;

    double3 sum_s = sm_helper[s].sum_app;
    double3 sum_k = sm_helper[k].sum_app;
    double3 sum_f;
    sum_f.x = sum_s.x + sum_k.x;
    sum_f.y = sum_s.y + sum_k.y;
    sum_f.z = sum_s.z + sum_k.z;

    double3 sq_sum_s = sm_helper[s].sq_sum_app;
    double3 sq_sum_k = sm_helper[k].sq_sum_app;
    double3 sq_sum_f;
    sq_sum_f.x = sq_sum_s.x + sq_sum_k.x;
    sq_sum_f.y = sq_sum_s.y + sq_sum_k.y;
    sq_sum_f.z = sq_sum_s.z + sq_sum_k.z;

    /************************************************

        Data Term [Standard BASS]

    ************************************************/



    /************************************************

        Proper Data Term [Fixed Var; Standard BASS+]

    ************************************************/

    // -- marginal likelihood --
    // double lprob_k = marginal_likelihood_app(sum_k,sq_sum_k,count_k,sigma2_app);
    // double lprob_s = marginal_likelihood_app(sum_s,sq_sum_s,count_s,sigma2_app);
    // double lprob_f = marginal_likelihood_app(sum_f,sq_sum_f,count_f,sigma2_app);


    /************************************************

       Appearance Variance Minimize; just for dev

    ************************************************/

    double sigma2_prior_var = 1.;
    double lprob_k = appearance_variance(sum_k,sq_sum_k,count_k,sigma2_prior_var);
    double lprob_s = appearance_variance(sum_s,sq_sum_s,count_s,sigma2_prior_var);
    double lprob_f = appearance_variance(sum_f,sq_sum_f,count_f,sigma2_prior_var);


    // -- include size term --
    // int sp_size2 = sp_size*sp_size;
    lprob_k += size_likelihood(count_k,sp_size,sigma2_size);
    lprob_s += size_likelihood(count_s,sp_size,sigma2_size);
    lprob_f += size_likelihood(count_f,sp_size,sigma2_size);

    // -- include size term --
    // lprob_k += size_beta_likelihood(count_k,sp_size,sigma2_size,npix);
    // lprob_s += size_beta_likelihood(count_s,sp_size,sigma2_size,npix);
    // lprob_f += size_beta_likelihood(count_f,sp_size,sigma2_size,npix);

    // -- write --
    sm_helper[k].numerator_app = lprob_k;
    sm_helper[s].numerator_app = lprob_s;
    sm_helper[k].numerator_f_app = lprob_f;




}

__device__ double size_likelihood(int curr_count, int tgt_count, double sigma2) {
  double delta = 1.*(sqrt(1.*curr_count) - tgt_count);
  double lprob = - log(2*M_PI*sigma2)/2. - delta*delta/(2*sigma2);
  return lprob;
}

__device__ double size_beta_likelihood(int _count, int _tgt_count,
                                       double alpha, const int _npix) {
  if (alpha < 0){ return 0; }
  double count = 1.*_count;
  double npix = 1.*_npix;
  double tgt_count = 1*_tgt_count*_tgt_count;
  double beta = alpha*(npix-tgt_count)/(tgt_count+1e-10); // just in case...
  // double beta = alpha;
  double lprob = (alpha-1)*log(count/npix) + (beta-1)*log(1-count/npix);
  // lprob += lgammaf(npix*alpha/tgt_count) - lgammaf(alpha) - lgammaf(beta);
  lprob += lgamma(alpha+beta) - lgamma(alpha) - lgamma(beta);
  return lprob;
}

__device__ double appearance_variance(double3 sum_obs,double3 sq_sum_obs,
                                      int _num_obs, double sigma2) {
  double num_obs = 1.*_num_obs;
  double sample_var = (sq_sum_obs.x  - sum_obs.x*sum_obs.x);
  sample_var += (sq_sum_obs.y  - sum_obs.y*sum_obs.y);
  sample_var += (sq_sum_obs.z  - sum_obs.z*sum_obs.z);
  sample_var = sample_var/(3.*num_obs); // estimate sigma2
  // sample_var = sample_var/3.; // estimate sigma2
  double lprob = -sample_var/sigma2;
  return lprob;
}

__device__ double marginal_likelihood_app(double3 sum_obs,double3 sq_sum_obs,
                                          int _num_obs, double sigma2) {
  // ref: from https://www.cs.ubc.ca/~murphyk/Papers/bayesGauss.pdf
  // Equation 55 with modifications from Eq 57 where kappa = 1
  // -- silly; we should just replace forumla with tau2 -> infty limit --
  double tau2 = 1000.; // ~= mean has 95% prob to be within (-1,1)
  float num_obs = (float)_num_obs;

  // float3 mu_prior;

  double lprob_num = 1/2. * log(sigma2) - num_obs/2.0 * log(2*M_PI*sigma2) \
    - log(num_obs * tau2 + sigma2)/2.;
  double denom = 2*(num_obs*tau2+sigma2);
  double3 lprob;
  lprob.x = lprob_num - sq_sum_obs.x/(2*sigma2) \
    + tau2*sum_obs.x*sum_obs.x/(sigma2*denom);
  lprob.y = lprob_num - sq_sum_obs.y/(2*sigma2)
    + tau2*sum_obs.y*sum_obs.y/(sigma2*denom);
  lprob.z = lprob_num - sq_sum_obs.z/(2*sigma2)
    + tau2*sum_obs.z*sum_obs.z/(sigma2*denom);

  double _lprob;
  _lprob = lprob.x+lprob.y+lprob.z;
  return _lprob;
}



__global__ void calc_merge_candidate(int* seg, bool* border, int* sm_pairs,
                                     const int npix, const int nbatch,
                                     const int width, const int height,
                                     const int direction){
  // todo: add nbatch
    int idx = threadIdx.x + blockIdx.x * blockDim.x;  
    if (idx>=npix) return; 
    if(!border[idx]) return;
    int x = idx % width;
    int y = idx / width;

    int C = seg[idx]; // center 
    int W; // north, south, east,west            
    W = OUT_OF_BOUNDS_LABEL; // init 

    if(direction==1){
      // if ((y>0) && (y< height-1))
      if ((y>=0) && (y< (height-1)))
        {
          W = __ldg(&seg[idx+width]);  // down
        }
    }else{
      // if ((x>0) && (x< width-1))
      if ((x>0) && (x< width))
        {
          W = __ldg(&seg[idx-1]);  // left
        }
    }
        
    // If the nbr is different from the central pixel and is not out-of-bounds,
    // then it is a border pixel.
    if (W>=0 && C!=W){
      atomicMax(&sm_pairs[2*C+1],W);
    }

    return;        
}

__global__
void calc_split_candidate(int* dists, int* spix, bool* border,
                          int distance, int* done_flag, const int npix,
                          const int nbatch, const int width, const int height){
  

    // todo: add batch -- no nftrs
    int idx = threadIdx.x + blockIdx.x * blockDim.x;  
    if (idx>=npix) return; 
    int x = idx % width;
    int y = idx / width;
    int C = dists[idx]; // center 
    int spixC = spix[idx];
    // if (border[idx]) return; 

    if(C!=distance) return;

    if ((y>0)&&(idx-width>=0)){
      if((dists[idx-width]==-1) and (spix[idx-width] == spixC)){
        dists[idx-width] = distance+1;
        done_flag[0] = 1;
      }
    }          
    if ((x>0)&&(idx-1>=0)){
      if((dists[idx-1]==-1) and (spix[idx-1] == spixC)){
        dists[idx-1] = distance+1;
        done_flag[0] = 1;
      }
    }
    if ((y<height-1)&&(idx+width<npix)){
      if((dists[idx+width]==-1) and (spix[idx+width] == spixC)){
        dists[idx+width] = distance+1;
        done_flag[0] = 1;
      }
    }   
    if ((x<width-1)&&(idx+1<npix)){
      if((dists[idx+1]==-1) and (spix[idx+1] == spixC)){
        dists[idx+1] = distance+1;
        done_flag[0] = 1;
      }
    }
    
    return;        
}


__global__ void init_split(const bool* border, int* seg_gpu,
                           spix_params* sp_params,
                           const int nspix_buffer,
                           const int nbatch, const int width,
                           const int height, const int direction,
                           const int* seg, int* max_sp, int max_spix) {
  // only splits superpixel if the center of the superpixel is actually
  // where the superpixel lives.... since we use "mu_shape" 
  // however, this doesn't effect anything.

    // todo: add batch -- no nftrs
	int k = threadIdx.x + blockIdx.x * blockDim.x;  // the label
    // *max_sp = max_spix+1;
    *max_sp = max_spix; // MAX number -> MAX label
	if (k>=nspix_buffer) return;
	if (sp_params[k].valid == 0) return;
    int x;
    int y;
    if((direction==1)||(direction==-1))
    {
        x = int(sp_params[k].mu_shape.x)+direction;
        y = int(sp_params[k].mu_shape.y);
    }
    else
    {
        x = int(sp_params[k].mu_shape.x);
        y = int(sp_params[k].mu_shape.y)+direction;
    }
    
    int ind = y*width+x;
    // if (k <= max_spix){
    //   printf("k,ind,direction,width,mu_shape.x,mu_shape.y: %d,%d,%d,%d,%lf,%lf\n",
    //          k,ind,direction,width,sp_params[k].mu_shape.x,sp_params[k].mu_shape.y);
    // }
    if((ind<0)||(ind>width*height-1)) return;
    
    // printf("seg[ind]: %d\n",seg[ind]);
    // if(border[ind]) return;
    if (seg[ind]!=k) return;
    seg_gpu[ind] = 1;

}


__global__ void calc_seg_split(int* sm_seg1, int* sm_seg2, int* seg,
                               const int npix, int nbatch, int max_spix) {
  // todo -- nbatch
    int t = threadIdx.x + blockIdx.x * blockDim.x;
	if (t>=npix) return;
    int seg_val = __ldg(&seg[t]);

    // printf("[%d] (%d,%d)\n",t,sm_seg1[t],sm_seg2[t]);
    if(sm_seg1[t]>__ldg(&sm_seg2[t])) seg_val += (max_spix+1); 
    sm_seg1[t] = seg_val;

    return;
}


__global__ void sum_by_label(const float* img, const int* seg,
                                   spix_params* sp_params,
                                   spix_helper_sm* sm_helper,
                                   const int npix, const int nbatch,
                                   const int width, const int nftrs) {

	// getting the index of the pixel
    int t = threadIdx.x + blockIdx.x * blockDim.x;
	if (t>=npix) return;

	//get the label
    
	int k = __ldg(&seg[t]);
    float l = __ldg(& img[3*t]);
    float a = __ldg(& img[3*t+1]);
    float b = __ldg(& img[3*t+2]);
	atomicAdd(&sm_helper[k].count, 1); 
	atomicAdd(&sm_helper[k].sq_sum_app.x, l*l);
	atomicAdd(&sm_helper[k].sq_sum_app.y, a*a);
	atomicAdd(&sm_helper[k].sq_sum_app.z,b*b);
    atomicAdd(&sm_helper[k].sum_app.x, l);
	atomicAdd(&sm_helper[k].sum_app.y, a);
	atomicAdd(&sm_helper[k].sum_app.z, b);
    
	int x = t % width;
	int y = t / width; 
	atomicAdd((unsigned long long *)&sm_helper[k].sum_shape.x, x);
	atomicAdd((unsigned long long *)&sm_helper[k].sum_shape.y, y);
    return;
}





__global__
void split_hastings_ratio(const float* img, int* sm_pairs,
                          spix_params* sp_params,
                          spix_helper* sp_helper,
                          spix_helper_sm* sm_helper,
                          const int npix, const int nbatch,
                          const int width, const int nftrs,
                          const int nspix_buffer,
                          int sp_size, float log_alpha,
                          int max_spix, int* max_sp) {
  // todo -- add nbatch and nftrs
	// getting the index of the pixel
	int k = threadIdx.x + blockIdx.x * blockDim.x;  // the label

	if (k>=nspix_buffer) return;
	if (sp_params[k].valid == 0) return;
    
    int s = k + (max_spix + 1);
    if(s>=nspix_buffer) return;
    // float count_f = __ldg(&sp_params[k].count);
    float count_k = __ldg(&sm_helper[k].count);
    float count_s = __ldg(&sm_helper[s].count);
    float count_f = count_k + count_s;
    // printf("[%d]: %f,%f,%f\n",k,count_k,count_s,count_f);
    if((count_f<1)||(count_k<1)||(count_s<1)) return;

    float lprob_k = __ldg(&sm_helper[k].numerator_app);
    float lprob_s = __ldg(&sm_helper[s].numerator_app);
    float lprob_f = __ldg(&sm_helper[k].numerator_f_app);

    // -- compute hastings [old] --
    // float log_nominator = log_alpha\
    //   + lgammaf(count_k) +  lgammaf(count_s) + lprob_k + lprob_s;
    // float log_denominator = lgammaf(count_f) + lprob_f;
    // sm_helper[k].hasting = log_nominator - log_denominator;

    // -- compute hastings --
    double log_const = lgammaf(count_k) +  lgammaf(count_s) - lgammaf(count_f);
    // double log_const = 0;
    // double hastings = log_const + log_alpha + lprob_k + lprob_s - lprob_f;
    double hastings = log_alpha + lprob_k + lprob_s - lprob_f;
    sm_helper[k].hasting = hastings;
    sm_helper[k].merge = (sm_helper[k].hasting > 0);
    sm_helper[s].merge = (sm_helper[k].hasting > 0);
    // printf("info[%d,%d] %lf,%f,%f,%f,%lf\n",
    //        k,s,log_const,lprob_f,lprob_k,lprob_s,hastings);

    if((sm_helper[k].merge)) // split step
      {

        // printf("info[%d,%d] %lf,%f,%f,%f\n",k,s,log_const,lprob_f,lprob_k,lprob_s);
        s = atomicAdd(max_sp,1)+1; // ? can't multiple splits happen at one time? yes :D
        sm_pairs[2*k] = s;

        // -- update shape prior --
        float prior_count = max(sp_params[k].prior_count/2.0,8.0);
        sp_params[k].prior_count = prior_count;
        sp_params[s].prior_count = prior_count;

        // -- record number of splits/merges --
        // sp_params[k].sm_count -= 1;
        // sp_params[s].sm_count = sp_params[k].sm_count;

        // sp_params[k].prior_sigma_shape.x/=2;
        // sp_params[k].prior_sigma_shape.y/=2;
        // sp_params[k].prior_sigma_shape.z/=2;

        // double3 prior_sigma_shape;
        // prior_sigma_shape.x = 1./sp_size;
        // prior_sigma_shape.y = 0;
        // prior_sigma_shape.z = 1./sp_size;
        // sp_params[s].prior_sigma_shape = prior_sigma_shape;
        // sp_params[k].prior_sigma_shape = prior_sigma_shape;
        
        // double2 prior_mu_shape;
        // prior_mu_shape.x = 0;
        // prior_mu_shape.y = 0;
        // sp_params[s].prior_mu_shape = prior_mu_shape;
        // sp_params[s].prior_mu_shape_count = 1;
        // sp_params[s].prior_count =  sp_params[k].prior_count; 
        // sp_params[s].prior_sigma_shape = sp_params[k].prior_sigma_shape;
        

      }

}

__global__ void merge_sp(int* seg, bool* border, int* sm_pairs,
                         spix_params* sp_params,
                         spix_helper_sm* sm_helper,
                         const int npix, const int nbatch,
                         const int width, const int height){   
    int idx = threadIdx.x + blockIdx.x * blockDim.x;  
    if (idx>=npix) return; 
    int k = seg[idx]; // center 
    //if (sp_params[k].valid == 0) return;
    int f = sm_pairs[2*k+1];
    if(sm_helper[k].remove){
      seg[idx] =  f;
    }

    return;  
      
}

__global__ void split_sp(int* seg, int* sm_seg1, int* sm_pairs,
                         spix_params* sp_params,
                         spix_helper_sm* sm_helper,
                         const int npix, const int nbatch,
                         const int width, const int height, int max_spix){   

  // todo: add nbatch, no sftrs
    int idx = threadIdx.x + blockIdx.x * blockDim.x;  
    if (idx>=npix) return; 
    int k = seg[idx]; // center 
    int k2 = k + (max_spix + 1);
    if (sp_params[k].valid == 0){ return; }
    if ((sm_helper[k].merge == false)||sm_helper[k2].merge == false){
      return;
    }

    int s = sm_pairs[2*k];
    if (s < 0){ return; }
    if(sm_seg1[idx]==k2) seg[idx] = s;
    //seg[idx] = sm_seg1[idx];
    //printf("Add the following: %d - %d'\n", k,sm_pairs[2*k]);
    sp_params[s].valid = 1;
    // sp_params[sm_pairs[2*k]].prior_count = sp_params[sm_pairs[2*k]].prior_count;
    // sp_params[k].prior_sigma_shape.x = count*count;
    // sp_params[k].prior_sigma_shape.z = count*count;

    // ?

    return;  
}



__global__ void remove_sp(int* sm_pairs, spix_params* sp_params,
                          spix_helper_sm* sm_helper,
                          const int nspix_buffer) {

	// -- getting the index of the pixel --
	int k = threadIdx.x + blockIdx.x * blockDim.x;  // the label
	if (k>=nspix_buffer) return;
    int s = sm_pairs[2*k+1];
    if(s<0) return;
    bool is_cycle = sm_pairs[2*s+1] == k;
    if ((sp_params[k].valid == 0)||(sp_params[s].valid == 0)) return;    
    // if ((sm_helper[k].merge == true) && (sm_helper[f].merge == false) && (split_merge_pairs[2*f]==k) )
    // if ((sm_helper[k].merge==true)&&(sm_helper[s].merge==false)&&(sm_pairs[2*s]==k))
    // if ((sm_helper[k].merge==true)&&(sm_helper[s].merge==false))
    if((sm_helper[k].merge==true)&&((sm_helper[s].merge==false)||(is_cycle and(k<s))))
    // if ((sm_helper[k].merge == true) && (sm_helper[s].merge == false))
      {
        sm_helper[k].remove=true;
        sp_params[k].valid = 0;

        // -- update priors --
        sp_params[s].prior_count =sp_params[k].prior_count+sp_params[s].prior_count;

        // -- update number of splits/merges --
        // sp_params[k].sm_count += 1;
        // sp_params[s].sm_count += abs(sp_params[k].sm_count);
        // sp_params[s].sm_count += sp_params[k].sm_count;
        // sp_params[s].sm_count += 1;

        // double3 prior_sigma_shape;
        // prior_sigma_shape.x = 1./sp_size;
        // prior_sigma_shape.y = 0;
        // prior_sigma_shape.z = 1./sp_size;
        // sp_params[s].prior_sigma_shape.x = 1.0/sp_params[s].prior_count;
        // sp_params[s].prior_sigma_shape.y = 0.;
        // sp_params[s].prior_sigma_shape.z = 1.0/sp_params[s].prior_count;

      }
    else
      {
        sm_helper[k].remove=false;
      }
    
    return;
    
}


