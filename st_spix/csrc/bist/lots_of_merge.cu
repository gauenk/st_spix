#include "hip/hip_runtime.h"

/********************************************************************

      Run BASS using the propograted superpixel segs and params

********************************************************************/

// -- cpp imports --
#include <stdio.h>
#include <assert.h>

// -- thrust --
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/unique.h>
#include <thrust/sort.h>

// -- "external" import --
#include "structs.h"

// -- utils --
// #include "rgb2lab.h"
// #include "sparams_io.h"
#include "seg_utils.h"
#include "init_utils.h"
#include "init_seg.h"
#include "init_sparams.h"
#include "compact_spix.h"
#include "sparams_io.h"

// -- primary functions --
#include "split_merge_orig.h"
#include "update_params.h"
// #include "update_seg.h"

#define THREADS_PER_BLOCK 512


/**********************************************************

             -=-=-=-=- Main Function -=-=-=-=-=-

***********************************************************/

__host__ int lots_of_merge(float* img, int* seg,spix_params* sp_params,bool* border,
                  spix_helper* sp_helper,spix_helper_sm* sm_helper,
                  int* sm_seg1 ,int* sm_seg2, int* sm_pairs,
                  int niters, int niters_seg, int sm_start,
                  float sigma2_app,  float sigma2_size, int sp_size,
                  float potts, float alpha_hastings, float split_alpha, int nspix,
                  int nspix_buffer, int nbatch, int width, int height, int nftrs){

    // // -- init --
    int count = 1;
    int npix = height * width;
    int max_spix = nspix-1;

    // printf(".\n");
    // std::cout << "height, width: " << height << ", " << width << std::endl;
    float alpha_merge = 1.; // control the number of spix
    
    update_params(img, seg, sp_params, sp_helper, sigma2_app,
                  npix, sp_size, nspix_buffer, nbatch, width, nftrs);

    for (int idx = 0; idx < niters; idx++) {
      
      // -- Update Parameters --
      int _idx = 2;
      run_merge_orig(img, seg, border, sp_params,
                     sp_helper, sm_helper, sm_seg1, sm_seg2, sm_pairs,
                     alpha_hastings, alpha_merge, sigma2_app, sigma2_size, count, _idx,
                     max_spix,sp_size,npix,nbatch,width,height,nftrs,nspix_buffer);

      // -- Update Parameters --
      update_params(img, seg, sp_params, sp_helper, sigma2_app,
                        npix, sp_size, nspix_buffer, nbatch, width, nftrs);
      
    }

    update_params(img, seg, sp_params, sp_helper, sigma2_app,
                  npix, sp_size, nspix_buffer, nbatch, width, nftrs);
    store_sample_sigma_shape(sp_params,sp_helper,sp_size, nspix_buffer);

    CudaFindBorderPixels_end(seg, border, npix, nbatch, width, height);
    return max_spix;

}


/**********************************************************

             -=-=-=-=- Python API  -=-=-=-=-=-

***********************************************************/

std::tuple<int*,bool*,SuperpixelParams*>
// std::tuple<int*,bool*>
run_lots_of_merge(float* img, int* in_spix, int nspix,
         int nbatch, int height, int width, int nftrs,
         int niters, int niters_seg, int sm_start, int sp_size,
         float sigma2_app, float sigma2_size, float potts,
         float alpha_hastings, float split_alpha){


    // -- unpack --
    int npix = height*width;
    assert(nbatch==1);    

    
    // -- allocate filled spix --
    int* _spix = (int*)easy_allocate(nbatch*npix,sizeof(int));
    hipMemcpy(_spix,in_spix,nbatch*npix*sizeof(int),hipMemcpyDeviceToDevice);
    thrust::device_ptr<int> _spix_ptr = thrust::device_pointer_cast(_spix);
    thrust::device_vector<int> spix(_spix_ptr, _spix_ptr + npix);

    // -- init superpixels --
    // int nspix = init_seg(_spix,sp_size,width,height,nbatch);
    // printf("nspix: %d\n",nspix);

    // -- get min,max --
    // print_min_max(_spix, npix);

    // -- allocate memory --
    int nspix_buffer = nspix*30;
    const int sparam_size = sizeof(spix_params);
    const int helper_size = sizeof(spix_helper);
    bool* border = (bool*)easy_allocate(nbatch*npix,sizeof(bool));
    spix_params* sp_params=(spix_params*)easy_allocate(nspix_buffer,sparam_size);
    spix_helper* sp_helper=(spix_helper*)easy_allocate(nspix_buffer,helper_size);

    // -- INFO --
    thrust::device_vector<int> prop_ids0 = extract_unique_ids(_spix, npix, 0);
    nspix = compactify_new_superpixels(_spix,sp_params,prop_ids0,0,nspix,npix);
    printf("[lots of merge] input nspix: %d\n",nspix);
    // print_min_max(_spix, npix);
    // gpuErrchk( hipPeekAtLastError() );
    // gpuErrchk( hipDeviceSynchronize() );

    // -- allocate larger memory for prior map --
    int* sm_seg1 = (int*)easy_allocate(npix,sizeof(int));
    int* sm_seg2 = (int*)easy_allocate(npix,sizeof(int));
    int* sm_pairs = (int*)easy_allocate(2*npix,sizeof(int));
    const int sm_helper_size = sizeof(spix_helper_sm);
    spix_helper_sm* sm_helper=(spix_helper_sm*)easy_allocate(nspix_buffer,sm_helper_size);

    // -=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-
    //
    //                 Run BASS
    //
    // -=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-

    // gpuErrchk( hipPeekAtLastError() );
    // gpuErrchk( hipDeviceSynchronize() );
    // printf("hey\n");

    // -- init spix_params --
    mark_active_contiguous(sp_params,nspix,nspix_buffer,sp_size);
    // gpuErrchk( hipPeekAtLastError() );
    // gpuErrchk( hipDeviceSynchronize() );

    init_sp_params(sp_params,sigma2_app,img,_spix,sp_helper,
                   npix,nspix,nspix_buffer,nbatch,width,nftrs,sp_size);
    // gpuErrchk( hipPeekAtLastError() );
    // gpuErrchk( hipDeviceSynchronize() );

    CudaFindBorderPixels(_spix,border,npix,nbatch,width,height);

    // gpuErrchk( hipPeekAtLastError() );
    // gpuErrchk( hipDeviceSynchronize() );
    // printf("yay.\n");

    // -- run method --
    int max_spix = lots_of_merge(img, _spix, sp_params,
                        border, sp_helper, sm_helper, sm_seg1, sm_seg2, sm_pairs,
                        niters, niters_seg, sm_start, sigma2_app, sigma2_size,
                        sp_size, potts, alpha_hastings, split_alpha, nspix, nspix_buffer,
                        nbatch, width, height, nftrs);
    // print_min_max(_spix, npix);

    // int max_spix = nspix-1;
    // // fprintf(stdout,"[before] max_spix: %d\n",max_spix);

    // // -- view --
    // thrust::device_vector<int> uniq_spix(_spix_ptr, _spix_ptr + npix);
    // thrust::sort(uniq_spix.begin(),uniq_spix.end());
    // auto uniq_end = thrust::unique(uniq_spix.begin(),uniq_spix.end());
    // uniq_spix.erase(uniq_end, uniq_spix.end());
    // uniq_spix.resize(uniq_end - uniq_spix.begin());
    // printf("delta: %d\n",uniq_end - uniq_spix.begin());
    // int nactive = uniq_spix.size();
    // int* _uniq_spix = thrust::raw_pointer_cast(uniq_spix.data());
    // printf("nactive: %d\n",nactive);
    // int _num_blocks = ceil( double(nactive) / double(THREADS_PER_BLOCK) ); 
    // dim3 _nblocks(_num_blocks);
    // dim3 _nthreads(THREADS_PER_BLOCK);
    // _view_prior_counts_kernel<<<_nblocks,_nthreads>>>(sp_params, _uniq_spix, nactive);

    // -- only keep superpixels which are alive --
    thrust::device_vector<int> prop_ids = extract_unique_ids(_spix, npix, 0);
    nspix = compactify_new_superpixels(_spix,sp_params,prop_ids,0,max_spix,npix);
    printf("[lots of merge] nspix: %d\n",nspix);
    // print_min_max(_spix, npix);

    // -- get spixel parameters as tensors --
    thrust::copy(_spix_ptr,_spix_ptr+npix,spix.begin());
    thrust::device_vector<int> uniq_ids = get_unique(_spix,npix);
    int num_ids = uniq_ids.size();
    int* _uniq_ids = thrust::raw_pointer_cast(uniq_ids.data());
    SuperpixelParams* params = get_params_as_vectors(sp_params,_uniq_ids,num_ids,nspix);
    run_update_prior(params,_uniq_ids, npix, nspix, 0,false);
    // run_update_prior(params,_uniq_ids, npix, nspix, nspix_prev,false);
    CudaFindBorderPixels_end(_spix,border,npix,nbatch,width,height);


    // -- free --
    hipFree(sm_helper);
    hipFree(sm_pairs);
    hipFree(sm_seg2);
    hipFree(sm_seg1);
    hipFree(sp_helper);
    hipFree(sp_params);
    // hipFree(border);

    // -- return! --
    return std::make_tuple(_spix,border,params);
    // return std::make_tuple(_spix,border);
}


