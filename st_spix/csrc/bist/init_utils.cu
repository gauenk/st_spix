/*************************************************

          This script helps allocate
          and initialize memory for
          supporting information

**************************************************/

#include "init_utils.h"
#include <thrust/system_error.h>
#include <thrust/system/cuda/error.h>
#define THREADS_PER_BLOCK 512

/*************************************************

               Allocation

**************************************************/

void throw_on_cuda_error_prop(hipError_t code){ // new name since two .so objects (ugh)
  if(code != hipSuccess){
    throw thrust::system_error(code, thrust::cuda_category());
  }
}

void* easy_allocate(int size, int esize){
  void* mem;
  try {
    throw_on_cuda_error_prop(hipMalloc((void**)&mem,size*esize));
  }
  catch (thrust::system_error& e) {
    std::cerr << "CUDA error after hipMalloc: " << e.what() << std::endl;
  }
  return mem;
}

void* easy_allocate_cpu(int size, int esize){
  void* mem;
  mem = malloc(size*esize);
  if (!mem) {
    throw "Malloc Failed.";
  }
  return mem;
}

