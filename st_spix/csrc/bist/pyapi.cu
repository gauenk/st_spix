#include "hip/hip_runtime.h"

// -- pytorch api --
#include <torch/extension.h>
#include <torch/types.h>
#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)
#define THREADS_PER_BLOCK 512

// -- basic --
#include <iostream>
#include <fstream>
#include <dirent.h>
#include <unistd.h> // For access()


// -- thrust --
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/extrema.h>
#include <thrust/unique.h>
#include <thrust/sort.h>

// // -- opencv --
// #include <opencv2/opencv.hpp>

// -- local --
// #include "file_io.h"
#include "structs.h"
#include "init_utils.h"
#include "rgb2lab.h"
#include "bass.h"
#include "prop.h"
#include "shift_and_fill.h"
#include "seg_utils.h" // dev only
// #include "update_seg.h" // dev only
#include "split_disconnected.h"

// -- demo --
// #include "demo_utils.h"

// using namespace cv;
using namespace std;


torch::Tensor main_loop(torch::Tensor vid, torch::Tensor flows,
                        int niters, int sp_size, float sigma2_app,
                        float potts, float alpha, float split_alpha,
                        int target_nspix, bool video_mode, bool rgb2lab_b){

  // -- viz inputs --
  // printf("niters: %d, sp_size: %d, sigma2_app: %.3f, potts: %.3f, alpha: %.3f, split_alpha: %.3f, target_nspix: %d, video_mode: %s\n",niters, sp_size, sigma2_app, potts, alpha, split_alpha, target_nspix, video_mode ? "true" : "false");  

  // -- unpack shape --
  int nframes = vid.size(0);
  int height = vid.size(1);
  int width = vid.size(2);
  int nftrs = vid.size(3);
  int npix = height*width;
  int nbatch = 1;

  // -- legacy --
  int sm_start = 0;
  float sigma2_size = 0.0;

  // -- actually, not an input --
  int niters_seg = 4;
  // float split_alpha = 0.0;
  float merge_alpha = 0.0;

  // -- not controlled in python --
  float thresh_relabel = 1e-5;
  float thresh_new = 1e-2;

  // -- alloc options --
  auto options_f32 = torch::TensorOptions().dtype(torch::kFloat32)
    .layout(torch::kStrided).device(vid.device());
  auto options_i32 = torch::TensorOptions().dtype(torch::kInt32)
    .layout(torch::kStrided).device(vid.device());

  // -- allocate spix --
  torch::Tensor spix_th = torch::zeros({nframes, height, width}, options_i32);

  // -- init --
  // float* img_lab = (float*)easy_allocate(npix*3,sizeof(float));
  float* img_rgb;
  float* flow = nullptr;
  int* spix_prev = nullptr;
  SuperpixelParams* params_prev = nullptr;

  // -- start loop --
  for(int fidx=0; fidx < nframes; fidx++){
  

    // -- prepare images --
    img_rgb = vid[fidx].data_ptr<float>();
    // if (rgb2lab_b) {
    //   rgb2lab(img_rgb,img_lab,nbatch,npix); // convert image to LAB
    // }else {
    //   hipMemcpy(img_lab,img_rgb,npix*3,hipMemcpyDeviceToDevice);
    // }

    // -- unpack flow --
    if ((video_mode) and (fidx>0)){
      flow = flows[fidx-1].data_ptr<float>();
    }

    // -- init -- 
    int* spix = nullptr;
    bool* border = nullptr;
    int nspix = -1;
    SuperpixelParams* params = nullptr;

    if ((fidx == 0)||(video_mode == false)){
      // -- single image --
      auto out = run_bass(img_rgb, nbatch, height, width, nftrs,
                          niters, niters_seg, sm_start,
                          sp_size,sigma2_app,sigma2_size,
                          potts,alpha,split_alpha,target_nspix);
      spix = std::get<0>(out);
      border = std::get<1>(out);
      params = std::get<2>(out);

      // -- fill --
      torch::Tensor _spix_th = torch::from_blob(spix, {height, width}, options_i32);
      spix_th.index_put_({fidx}, _spix_th);

      // -- free data --
      // del _spix_th;
      hipFree(border);

    }else{

      // -- shift & fill --
      auto out_saf = shift_and_fill(spix_prev,params_prev,flow,
                                    nbatch,height,width);
      int* filled_spix = std::get<0>(out_saf);
      int* shifted_spix = std::get<1>(out_saf);

      // -- count percentage invalid --
      int ninvalid = count_invalid(shifted_spix,npix);
      float iperc = ninvalid / (1.0*npix);
      if (iperc > 0.20){
        niters = 12;
      }else if(iperc < 0.01){
        niters = 4;
      }else {
        niters = 8;
      }

      // -- propogate --
      auto out = run_prop(img_rgb, nbatch, height, width, nftrs,
                          niters, niters_seg, sm_start,
                          sp_size,sigma2_app,sigma2_size,
                          potts,alpha,filled_spix,shifted_spix,params_prev,
                          thresh_relabel, thresh_new,
                          merge_alpha, split_alpha, target_nspix);
      spix = std::get<0>(out);
      border = std::get<1>(out);
      params = std::get<2>(out);

      // -- fill --
      torch::Tensor _spix_th = torch::from_blob(spix, {height, width}, options_i32);
      spix_th.index_put_({fidx}, _spix_th);

      // -- free data --
      // del _spix_th;
      hipFree(border);

      // -- free --
      hipFree(filled_spix);
      hipFree(shifted_spix);

    }

    // -- [propogate info!] --
    if (fidx>0){
      hipFree(spix_prev);
      free(params_prev);
    }
    if (fidx == (nframes-1)){
      hipFree(spix);
      free(params);
    }else{
      spix_prev = spix;
      params_prev = params;
    }

  }
  // hipFree(img_lab);



  return spix_th;
}


// std::tuple<torch::Tensor,torch::Tensor>
torch::Tensor
bist_forward_cuda(const torch::Tensor vid, const torch::Tensor flows,
                  int niters, int sp_size, float potts,
                  float sigma2_app, float alpha, float split_alpha,
                  bool video_mode, bool rgb2lab_b){

  // -- check --
  CHECK_INPUT(vid);
  CHECK_INPUT(flows);

  int target_nspix = 0;
  auto out = main_loop(vid, flows, niters,  sp_size,
                       sigma2_app, potts, alpha, split_alpha,
                       target_nspix, video_mode, rgb2lab_b);

  return out;
}





__global__ void GetImageOverlaid(float* filled, float* image, float* color,
                                 const bool* border, const int npix, const int xdim){
  int t = threadIdx.x + blockIdx.x * blockDim.x;  
  if (t>=npix) return;
  t = t + npix*blockIdx.y; // offset via batch

  if (border[t]){
    // -- for a nice grey --
    // filled[3*t] = 50;
    // filled[3*t+1] = 50;
    // filled[3*t+2] = 50;
    // -- for a sharp blue --
    // filled[3*t] = 0.0;
    // filled[3*t+1] = 0;
    // filled[3*t+2] = 1.0;
    filled[3*t] = color[0];
    filled[3*t+1] = color[1];
    filled[3*t+2] = color[2];
  }else{
    filled[3*t] = max(min(image[3*t],1.),0.0);
    filled[3*t+1] = max(min(image[3*t+1],1.),0.0);
    filled[3*t+2] = max(min(image[3*t+2],1.),0.0);
  }
    
}


__host__ void
CUDA_get_image_overlaid(float* filled, float* image, float* color,
                        const bool* border, const int npix,
                        const int xdim, const int nbatch){
  int num_block = ceil( double(npix) / double(THREADS_PER_BLOCK) ); 
  dim3 ThreadPerBlock(THREADS_PER_BLOCK,1);
  dim3 BlockPerGrid(num_block,nbatch);
  GetImageOverlaid<<<BlockPerGrid,ThreadPerBlock>>>(filled, image, color,
                                                    border, npix, xdim);
}


torch::Tensor get_marked_video(torch::Tensor vid,
                               torch::Tensor spix, torch::Tensor color){
  
  // -- check --
  CHECK_INPUT(vid);
  CHECK_INPUT(spix);
  CHECK_CONTIGUOUS(color);

  // -- unpack shape --
  int nframes = vid.size(0);
  int height = vid.size(1);
  int width = vid.size(2);
  int nftrs = vid.size(3);
  int npix = height*width;
  int nbatch = 1;

  // -- manage color input --
  long long ncolors = at::numel(color);
  assert(nftrs==ncolors);
  color = color.to(vid.device());

  // -- alloc border and marked image --
  auto options_f32 = torch::TensorOptions().dtype(torch::kFloat32)
    .layout(torch::kStrided).device(vid.device());
  torch::Tensor marked = torch::zeros({nframes, height, width, nftrs}, options_f32);

  // -- unpack pointers --
  float* _vid = vid.data_ptr<float>();
  int* _spix = spix.data_ptr<int>();
  float* _color = color.data_ptr<float>();
  float* _marked = marked.data_ptr<float>();

  // -- get the border --
  bool* border = (bool*)easy_allocate(nframes*npix,sizeof(bool));
  CudaFindBorderPixels_end(_spix, border, npix, nframes, width, height);

  // -- fill with marked values --
  CUDA_get_image_overlaid(_marked, _vid, _color, border, npix, width, nframes);

  // -- free memory --
  hipFree(border);

  return marked;
}



void init_bist(py::module &m){
  m.def("bist_forward", &bist_forward_cuda,"BIST");
  m.def("get_marked_video", &get_marked_video,"get marked video");
  // m.def("bass_forward", &bass_forward_cuda,
  //       "BASS");
}

