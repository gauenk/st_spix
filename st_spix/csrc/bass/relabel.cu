#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <cuda/std/type_traits>

__device__  __host__
int midpoint(int a, int b)
{
    return a + (b-a)/2;
}

__device__ __host__
int eval(int A[], int i, int val, int imin, int imax)
{

    int low = (A[i] <= val);
    int high = (A[i+1] > val);

    if (low && high) {
        return 0;
    } else if (low) {
        return -1;
    } else {
        return 1;
    }
}

__device__ __host__
int binary_search(int A[], int val, int imin, int imax){
    while (imax >= imin) {
        int imid = midpoint(imin, imax);
        int e = eval(A, imid, val, imin, imax);
        if(e == 0) {
            return imid;
        } else if (e < 0) {
            imin = imid;
        } else {         
            imax = imid;
        }
    }

    return -1;
}



template<bool mode> __global__
void relabel_spix(int* spix, int* ids, int npix, int nspix){

  // -- filling superpixel params into image --
  extern __shared__ int buff[];
  int ix = threadIdx.x + blockIdx.x * blockDim.x;  
  if (ix>=npix) return; 

  // -- offset super pixels --
  int spix_ix = *(spix + ix);
  int new_id = -1;

  // -- offset of kx -- todo: replace with binary search
  for (int kx=0; kx<nspix; kx++){
    if (ids[kx] == spix_ix){
      new_id = kx;
      break;
    }
  }
  (spix + ix)[0] = new_id;
}



// -- templating --
template void __global__ relabel_spix<false>(int*, int*, int, int);
template void __global__ relabel_spix<true>(int*, int*, int, int);
