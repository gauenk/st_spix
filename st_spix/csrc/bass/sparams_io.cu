#include "hip/hip_runtime.h"

#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cmath>

#include "core/Superpixels.h"


__global__
void copy_spix_to_params(float* means, float* cov, int* counts,
                         superpixel_params* sp_params, int* ids, int nspix){

    // -- filling superpixel params into image --
    int ix = threadIdx.x + blockIdx.x * blockDim.x;  
    if (ix>=nspix) return; 

    // -- offset memory access --
    float* means_ix = means + ix * 5;
    float* cov_ix = cov + ix * 4;
    int* counts_ix = counts + ix;

    // -- read spix --
    int sp_index = ids[ix];
    if (sp_index < 0){ return; }
    auto params_ix = sp_params[sp_index];
      
    // -- fill params --
    cov_ix[0] = params_ix.sigma_s.x;
    cov_ix[1] = params_ix.sigma_s.y;
    cov_ix[2] = params_ix.sigma_s.z;
    cov_ix[3] = params_ix.logdet_Sigma_s;
    means_ix[0] = params_ix.mu_i.x;
    means_ix[1] = params_ix.mu_i.y;
    means_ix[2] = params_ix.mu_i.z;
    means_ix[3] = params_ix.mu_s.x;
    means_ix[4] = params_ix.mu_s.y;
    counts_ix[0] = params_ix.count;
}


__global__
void copy_spix_to_params_parents(float* means, float* cov,
                                 int* counts, int* spix_parents,
                                 superpixel_params* sp_params, int* ids, int nspix){

    // -- filling superpixel params into image --
    int ix = threadIdx.x + blockIdx.x * blockDim.x;  
    if (ix>=nspix) return; 

    // -- offset memory access --
    float* means_ix = means + ix * 5;
    float* cov_ix = cov + ix * 4;
    int* counts_ix = counts + ix;
    int* spix_ix = spix_parents + ix;

    // -- read spix --
    int sp_index = ids[ix];
    if (sp_index < 0){ return; }
    auto params_ix = sp_params[sp_index];
      
    // -- fill params --
    cov_ix[0] = params_ix.sigma_s.x;
    cov_ix[1] = params_ix.sigma_s.y;
    cov_ix[2] = params_ix.sigma_s.z;
    cov_ix[3] = params_ix.logdet_Sigma_s;
    means_ix[0] = params_ix.mu_i.x;
    means_ix[1] = params_ix.mu_i.y;
    means_ix[2] = params_ix.mu_i.z;
    means_ix[3] = params_ix.mu_s.x;
    means_ix[4] = params_ix.mu_s.y;
    counts_ix[0] = params_ix.count;
    spix_ix[0] = params_ix.parent_spix;
}


__global__
void copy_spix_to_params_icov2cov(float* means, float* cov, int* counts,
                                  superpixel_params* sp_params, int* ids, int nspix){

    // -- filling superpixel params into image --
    int ix = threadIdx.x + blockIdx.x * blockDim.x;  
    if (ix>=nspix) return; 

    // -- offset memory access --
    float* means_ix = means + ix * 5;
    float* cov_ix = cov + ix * 4;
    int* counts_ix = counts + ix;

    // -- read spix --
    int sp_index = ids[ix];
    if (sp_index < 0){ return; }
    auto params_ix = sp_params[sp_index];
      
    // -- fill params --
    cov_ix[0] = params_ix.sigma_s.x;
    cov_ix[1] = params_ix.sigma_s.y;
    cov_ix[2] = params_ix.sigma_s.z;
    cov_ix[3] = params_ix.logdet_Sigma_s;
    means_ix[0] = params_ix.mu_i.x;
    means_ix[1] = params_ix.mu_i.y;
    means_ix[2] = params_ix.mu_i.z;
    means_ix[3] = params_ix.mu_s.x;
    means_ix[4] = params_ix.mu_s.y;
    counts_ix[0] = params_ix.count;

    // -- invert cov --
    // .x => sx   .z => sy
    // .y => rho * sx * sy
    double inv_detC = exp(cov_ix[3]);
    cov_ix[0] = inv_detC * cov[2];
    cov_ix[1] = -inv_detC * cov[1];
    cov_ix[2] = inv_detC * cov[0];

}



__global__
void copy_params_to_spix(float* means, float* cov, int* counts,
                         superpixel_params* sp_params, int* ids, int nspix){

    // -- filling superpixel params into image --
    int ix = threadIdx.x + blockIdx.x * blockDim.x;  
    if (ix>=nspix) return; 

    // -- offset memory access --
    float* means_ix = means + ix * 5;
    float* cov_ix = cov + ix * 4;
    int* counts_ix = counts + ix;

    // -- read spix --
    int sp_index = ids[ix];
    if (sp_index < 0){ return; }
    auto params_ix = sp_params[sp_index];
      
    // -- fill params --
    double3 sigma_s;
    sigma_s.x = cov_ix[0];
    sigma_s.y = cov_ix[1];
    sigma_s.z = cov_ix[2];
    params_ix.sigma_s = sigma_s;
    // params_ix.sigma_s.x = cov_ix[0];
    // params_ix.sigma_s.y = cov_ix[1];
    // params_ix.sigma_s.z = cov_ix[2];
    params_ix.logdet_Sigma_s = cov_ix[3];

    float3 mu_i;
    mu_i.x = means_ix[0];
    mu_i.y = means_ix[1];
    mu_i.z = means_ix[2];
    params_ix.mu_i = mu_i;
    // params_ix.mu_i.x = means_ix[0];
    // params_ix.mu_i.y = means_ix[1];
    // params_ix.mu_i.z = means_ix[2];
    params_ix.mu_s.x = means_ix[3];
    params_ix.mu_s.y = means_ix[4];
    params_ix.count = counts_ix[0];
}


/*********************************************************

        Allocate Memory (Vid,Seg) -> (Params)

/*********************************************************/

superpixels_params* allocate_superpixels(int nspix_buffer){
  const int sofsparams = sizeof(superpixel_params);
  superpixel_params* params;
  try {
    throw_on_cuda_error(hipMalloc((void**)&sp_params, nspix_buffer * sofsparams));
  }
  catch (thrust::system_error& e) {
    std::cerr << "CUDA error after hipMalloc: " << e.what() << std::endl;
    hipSetDevice(0);
  }
  return params;
}

superpixels_GPU_helper_sm* allocate_helper(int nspix_buffer){
  const int sofsphelper_sm = sizeof(superpixel_GPU_helper_sm);
  superpixel_GPU_helper_sm* helper;
  try {
    throw_on_cuda_error(hipMalloc((void**)&helper,nspix_buffer*sofsphelper_sm));
  }
  catch (thrust::system_error& e) {
    std::cerr << "CUDA error after hipMalloc: " << e.what() << std::endl;
    hipSetDevice(0);
  }
  return helper;
}

// __host__ void update_param(const float* image_gpu_double, const int* seg_gpu,
//                            superpixel_params* sp_params,
//                            superpixel_GPU_helper* sp_gpu_helper,
//                            const int nPixels, const int nSps,
//                            const int nSps_buffer, const int nbatch,
//                            const int xdim, const int ydim, const int nftrs,
//                            const int prior_sigma_s, const int prior_count){

// // -- helper params --
// superpixels_params* allocate_superpixels(int nspix_buffer){



// const int sofsparams = sizeof(superpixel_params);
//     const int sofsphelper = sizeof(superpixel_GPU_helper);
//     const int sofsphelper_sm = sizeof(superpixel_GPU_helper_sm);
//     const int sofpost_changes = sizeof(post_changes_helper);

//     sp_params_cpu = (superpixel_params*)malloc(nSPs_buffer * sofsparams);

//     try {
//         throw_on_cuda_error(hipMalloc((void**)&sp_params, nSPs_buffer * sofsparams));
//         throw_on_cuda_error(hipMalloc((void**)&sp_gpu_helper,nSPs_buffer*sofsphelper));
//         throw_on_cuda_error(hipMalloc((void**)&sp_gpu_helper_sm,\
//                                        nSPs_buffer*sofsphelper_sm));
//         throw_on_cuda_error(hipMalloc((void**)&post_changes, nPixels * sofpost_changes));
//     }
//     catch (thrust::system_error& e) {
//         std::cerr << "CUDA error after hipMalloc: " << e.what() << std::endl;
//         hipSetDevice(0);
//     }

