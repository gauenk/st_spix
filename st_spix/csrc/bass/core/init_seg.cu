#include "hip/hip_runtime.h"
#include <string>
#include <iostream>
#include <fstream>
#include <sstream>
#include <float.h>
#include <filesystem>
using namespace std;

#define THREADS_PER_BLOCK 512

#include "../share/gpu_utils.h"
#include "init_seg.h"

#include <stdio.h>

__host__ int CudaInitSeg(int* seg_cpu, int* seg_gpu, int* split_merge_pairs, int nPts,
                         int sz, int nbatch, int xdim, int ydim, bool use_hex){	
  // -- todo; add nbatch --

  	dim3 ThreadPerBlock(THREADS_PER_BLOCK,1);
    use_hex = true;
    int num_block_pixel = ceil(double(nPts+1) / double(THREADS_PER_BLOCK));
    dim3 BlockPerGrid_pixel(num_block_pixel,nbatch);
	if (!use_hex){
      InitSquareSeg<<<BlockPerGrid_pixel,ThreadPerBlock>>>(seg_gpu,nPts,sz,
                                                           nbatch, xdim, ydim);
        hipMemcpy(seg_cpu, seg_gpu, nPts*sizeof(int), hipMemcpyDeviceToHost);

	}else{
      std::stringstream batch_str, xdim_str, ydim_str, sz_str;
      batch_str << nbatch;
        xdim_str << xdim;
        ydim_str << ydim;
        sz_str << sz;     
        std::string root =  ".bass_cache";
        std::string file_path = root + "/" + batch_str.str() + "_" + xdim_str.str() \
          + "_" + ydim_str.str() + "_" + sz_str.str() + ".bin";

        // length of each side   
        double H = sqrt( double(pow(sz, 2)) / (1.5 *sqrt(3.0)) );
        double w = sqrt(3.0) * H;
        //printf("%1f \n", H);
        //printf("%1f \n", w);

        //calculate how many hexagons are on x and y axis
        int max_num_sp_x = (int) floor(double(xdim)/w) + 1;
        int max_num_sp_y = (int) floor(double(ydim)/(1.5*H)) + 1;

        int max_nSPs = max_num_sp_x * max_num_sp_y * 4; //Roy -Change

        //printf("%d \n", max_num_sp_x);
       // printf("%d \n", max_num_sp_y);
       // printf("%d \n", max_nSPs);

        // -- create directory --
        if (not std::filesystem::is_directory(root)){
          std::filesystem::create_directory(root);
        }

        if (loadArray(seg_cpu, nPts, file_path)){
            hipMemcpy(seg_gpu,seg_cpu,nPts*sizeof(int),hipMemcpyHostToDevice);
        }else{
        // if (true){

            int num_block_sp =  ceil(double(max_nSPs) /double(THREADS_PER_BLOCK));
            dim3 BlockPerGrid_sp(num_block_sp,nbatch);
            double* centers;
            hipMalloc((void**) &centers, 2*max_nSPs*sizeof(double));
            InitHexCenter<<<BlockPerGrid_sp,ThreadPerBlock>>>(centers, \
                              H, w, max_nSPs, max_num_sp_x, nbatch, xdim, ydim); 
            InitHexSeg<<<BlockPerGrid_pixel,ThreadPerBlock>>>(seg_gpu, \
                                            centers, max_nSPs, nPts, nbatch, xdim);
            hipFree(centers);
            hipMemcpy(seg_cpu, seg_gpu, nPts*sizeof(int), hipMemcpyDeviceToHost);
            // saveArray(seg_cpu, nPts, file_path);


            // // -- Remove Small Edges --
            // RemoveSmallEdges<<<BlockPerGrid_pixel,ThreadPerBlock>>>(seg_gpu, \
            //                                 centers, max_nSPs, nPts, nbatch, xdim);


        }
        
	}

    int nSPs = get_max(seg_cpu, nPts)+1;
	return nSPs;
}



// __global__ void RemoveSmallEdges(double* centers, double H, double w, int max_nPts,
//                                  int max_num_sp_x, int nbatch, int xdim, int ydim){

// }


__global__ void InitHexCenter(double* centers, double H, double w, int max_nPts,
                              int max_num_sp_x, int nbatch, int xdim, int ydim){
  // todo; add batch
	int idx = threadIdx.x + blockIdx.x * blockDim.x; 
	if (idx >= max_nPts) return;

    int x = idx % max_num_sp_x; 
    int y = idx / max_num_sp_x; 

    double xx = double(x) * w;
    double yy = double(y) * 1.5 *H; 
    
    if (y%2 == 0){
        xx = xx + 0.5*w;
    }
    
    centers[2*idx]  = xx;
    centers[2*idx+1]  = yy;    
}




__global__ void InitHexSeg(int* seg, double* centers, int K, int nPts,
                           int nbatch, int xdim){
  // todo ;add nbatch [just copy this rather than add nbatch...]
	int idx = threadIdx.x + blockIdx.x * blockDim.x; 	
	if (idx >= nPts) return;

    int x = idx % xdim;
    int y = idx / xdim;   

    double dx,dy,d2;
    double D2 = DBL_MAX; 
    for (int j=0; j < K;  j++){
        dx = (x - centers[j*2+0]);
        dy = (y - centers[j*2+1]);
        d2 = dx*dx + dy*dy;
        if ( d2 <= D2){
              D2 = d2;  
              seg[idx]=j+1;
        }           
    } 
    return;	
}



// for everypixel, assign it to a superptxel
__global__ void  InitSquareSeg(int* seg, int nPts, int sz,
                               int nbatch, int xdim, int ydim){
	int t = threadIdx.x + blockIdx.x * blockDim.x; 
	if (t>=nPts) return;
	
    float side_x = float(xdim)/float(ceil(float(xdim)/float(sz)));
    float side_y = float(ydim)/float(ceil(float(ydim)/float(sz)));
    //side_x += xdim/ side_x;
    //side_y += xdim/ side_x;

	//how many superpixels per col
	//int sp_y = (ydim%side_y<1)? int(ydim/side_y) : ( (int)floor(int(ydim/side_y)));

	int x = t % xdim;  
	int y =  t / xdim;

	//int i = (x%sz==0)? int(x/side_x): ((int) floor(x/side_x)); // which col
    //int j = (y%sz==0)? int(y/side_y): ((int) floor(y/side_y)); //which row
    int i = floor(float(x)/side_x);
    int j = floor(float(y)/side_y);
	seg[t] = j + i*float(ceil(float(ydim)/float(sz))) +1;  
}

